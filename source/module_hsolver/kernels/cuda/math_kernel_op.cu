#include "hip/hip_runtime.h"
#include "module_base/module_device/memory_op.h"
#include "module_hsolver/kernels/math_kernel_op.h"
#include "module_psi/psi.h"
#include "module_base/tool_quit.h"

#include <base/macros/macros.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/execution_policy.h>
#include <thrust/inner_product.h>

namespace hsolver
{
const int warp_size = 32;
// const unsigned int full_mask = 0xffffffff;
const int thread_per_block = 256;
}

template <>
struct GetTypeReal<thrust::complex<float>> {
    using type = float; /**< The return type specialization for std::complex<double>. */
};
template <>
struct GetTypeReal<thrust::complex<double>> {
    using type = double; /**< The return type specialization for std::complex<double>. */
};
namespace hsolver {
template <typename T>
struct GetTypeThrust {
    using type = T;
};

template <>
struct GetTypeThrust<std::complex<float>> {
    using type = thrust::complex<float>; /**< The return type specialization for std::complex<float>. */
};

template <>
struct GetTypeThrust<std::complex<double>> {
    using type = thrust::complex<double>; /**< The return type specialization for std::complex<float>. */
};

static hipblasHandle_t cublas_handle = nullptr;

static inline
void xdot_wrapper(const int &n, const float * x, const int &incx, const float * y, const int &incy, float &result) {
    cublasErrcheck(hipblasSdot(cublas_handle, n, x, incx, y, incy, &result));
}

static inline
void xdot_wrapper(const int &n, const double * x, const int &incx, const double * y, const int &incy, double &result) {
    cublasErrcheck(hipblasDdot(cublas_handle, n, x, incx, y, incy, &result));
}

void createGpuBlasHandle(){
    if (cublas_handle == nullptr) {
        cublasErrcheck(hipblasCreate(&cublas_handle));
    }
}

void destoryBLAShandle(){
    if (cublas_handle != nullptr) {
        cublasErrcheck(hipblasDestroy(cublas_handle));
        cublas_handle = nullptr;
    }
}

// template <typename FPTYPE>
// __forceinline__ __device__ void warp_reduce(FPTYPE& val) {
//     for (int offset = 16; offset > 0; offset >>= 1)
//         val += __shfl_down_sync(full_mask, val, offset);
// }

template <typename Real>
__global__ void line_minimize_with_block(
        thrust::complex<Real>* grad,
        thrust::complex<Real>* hgrad,
        thrust::complex<Real>* psi,
        thrust::complex<Real>* hpsi,
        const int n_basis,
        const int n_basis_max)
{
    int band_idx = blockIdx.x; // band_idx
    int tid = threadIdx.x; // basis_idx
    int item = 0;
    Real epsilo_0 = 0.0, epsilo_1 = 0.0, epsilo_2 = 0.0;
    Real theta = 0.0, cos_theta = 0.0, sin_theta = 0.0;
    __shared__ Real data[thread_per_block * 3];

    data[tid] = 0;

    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        data[tid] += (grad[item] * thrust::conj(grad[item])).real();
    }
    __syncthreads();
    // just do some parallel reduction in shared memory
    for (int ii = thread_per_block >> 1; ii > warp_size; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    // For threads in the same warp, it is better that they process the same work
    // Also, __syncwarp() should be used instead of __syncthreads()
    // Therefore we unroll the loop and ensure that the threads does the same work
    if (tid < warp_size) {
        data[tid] += data[tid + 32]; __syncwarp();
        data[tid] += data[tid + 16]; __syncwarp();
        data[tid] += data[tid + 8]; __syncwarp();
        data[tid] += data[tid + 4]; __syncwarp();
        data[tid] += data[tid + 2]; __syncwarp();
        data[tid] += data[tid + 1]; __syncwarp();
    }

    __syncthreads();

    Real norm = 1.0 / sqrt(data[0]);
    __syncthreads();

    data[tid] = 0;
    data[thread_per_block + tid] = 0;
    data[2 * thread_per_block + tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        grad[item] *= norm;
        hgrad[item] *= norm;
        data[tid] += (hpsi[item] * thrust::conj(psi[item])).real();
        data[thread_per_block + tid] += (grad[item] * thrust::conj(hpsi[item])).real();
        data[2 * thread_per_block + tid] += (grad[item] * thrust::conj(hgrad[item])).real();
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = thread_per_block >> 1; ii > warp_size; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
            data[thread_per_block + tid] += data[thread_per_block + tid + ii];
            data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + ii];
        }
        __syncthreads();
    }
    if (tid < warp_size) {
        data[tid] += data[tid + 32]; __syncwarp();
        data[tid] += data[tid + 16]; __syncwarp();
        data[tid] += data[tid + 8]; __syncwarp();
        data[tid] += data[tid + 4]; __syncwarp();
        data[tid] += data[tid + 2]; __syncwarp();
        data[tid] += data[tid + 1]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 32]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 16]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 8]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 4]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 2]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 1]; __syncwarp();
        data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + 32]; __syncwarp();
        data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + 16]; __syncwarp();
        data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + 8]; __syncwarp();
        data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + 4]; __syncwarp();
        data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + 2]; __syncwarp();
        data[2 * thread_per_block + tid] += data[2 * thread_per_block + tid + 1]; __syncwarp();
    }
    __syncthreads();
    epsilo_0 = data[0];
    epsilo_1 = data[thread_per_block];
    epsilo_2 = data[2 * thread_per_block];

    theta = 0.5 * abs(atan(2 * epsilo_1/(epsilo_0 - epsilo_2)));
    cos_theta = cos(theta);
    sin_theta = sin(theta);
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        psi [item] = psi [item] * cos_theta + grad [item] * sin_theta;
        hpsi[item] = hpsi[item] * cos_theta + hgrad[item] * sin_theta;
    }
}

template <typename Real>
__global__ void calc_grad_with_block(
        const Real* prec,
        Real* err,
        Real* beta,
        thrust::complex<Real>* psi,
        thrust::complex<Real>* hpsi,
        thrust::complex<Real>* grad,
        thrust::complex<Real>* grad_old,
        const int n_basis,
        const int n_basis_max)
{
    int band_idx = blockIdx.x; // band_idx
    int tid = threadIdx.x; // basis_idx
    int item = 0;
    Real err_st = 0.0;
    Real beta_st = 0.0;
    Real epsilo = 0.0;
    Real grad_2 = 0.0;
    thrust::complex<Real> grad_1 = {0, 0};
    __shared__ Real data[thread_per_block * 2];

    // Init shared memory
    data[tid] = 0;

    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        data[tid] += (psi[item] * thrust::conj(psi[item])).real();
    }
    __syncthreads();
    // just do some parallel reduction in shared memory
    for (int ii = thread_per_block >> 1; ii > warp_size; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    if (tid < warp_size) {
        data[tid] += data[tid + 32]; __syncwarp();
        data[tid] += data[tid + 16]; __syncwarp();
        data[tid] += data[tid + 8]; __syncwarp();
        data[tid] += data[tid + 4]; __syncwarp();
        data[tid] += data[tid + 2]; __syncwarp();
        data[tid] += data[tid + 1]; __syncwarp();
    }

    __syncthreads();

    Real norm = 1.0 / sqrt(data[0]);
    __syncthreads();

    data[tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        psi[item] *= norm;
        hpsi[item] *= norm;
        data[tid] += (hpsi[item] * thrust::conj(psi[item])).real();
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = thread_per_block >> 1; ii > warp_size; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
        }
        __syncthreads();
    }

    if (tid < warp_size) {
        data[tid] += data[tid + 32]; __syncwarp();
        data[tid] += data[tid + 16]; __syncwarp();
        data[tid] += data[tid + 8]; __syncwarp();
        data[tid] += data[tid + 4]; __syncwarp();
        data[tid] += data[tid + 2]; __syncwarp();
        data[tid] += data[tid + 1]; __syncwarp();
    }

    __syncthreads();
    epsilo = data[0];
    __syncthreads();

    data[tid] = 0;
    data[thread_per_block + tid] = 0;
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        grad_1 = hpsi[item] - epsilo * psi[item];
        grad_2 = thrust::norm(grad_1);
        data[tid] += grad_2;
        data[thread_per_block + tid] += grad_2 / prec[basis_idx];
    }
    __syncthreads();

    // just do some parallel reduction in shared memory
    for (int ii = thread_per_block >> 1; ii > warp_size; ii >>= 1) {
        if (tid < ii) {
            data[tid] += data[tid + ii];
            data[thread_per_block + tid] += data[thread_per_block + tid + ii];
        }
        __syncthreads();
    }

    if (tid < warp_size) {
        data[tid] += data[tid + 32]; __syncwarp();
        data[tid] += data[tid + 16]; __syncwarp();
        data[tid] += data[tid + 8]; __syncwarp();
        data[tid] += data[tid + 4]; __syncwarp();
        data[tid] += data[tid + 2]; __syncwarp();
        data[tid] += data[tid + 1]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 32]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 16]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 8]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 4]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 2]; __syncwarp();
        data[thread_per_block + tid] += data[thread_per_block + tid + 1]; __syncwarp();
    }

    __syncthreads();
    err_st = data[0];
    beta_st = data[thread_per_block];
    for (int basis_idx = tid; basis_idx < n_basis; basis_idx += thread_per_block) {
        item = band_idx * n_basis_max + basis_idx;
        grad_1 = hpsi[item] - epsilo * psi[item];
        grad[item] = -grad_1 / prec[basis_idx] + beta_st / beta[band_idx] * grad_old[item];
    }

    __syncthreads();
    if (tid == 0) {
        beta[band_idx] = beta_st;
        err[band_idx] = sqrt(err_st);
    }
}

// Define the CUDA kernel:
template <typename T>
__global__ void vector_div_constant_kernel(
    const int size,
    T* result,
    const T* vector,
    const typename GetTypeReal<T>::type constant)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        result[i] = vector[i] / constant;
    }
}

template <typename T>
__global__ void vector_mul_vector_kernel(
    const int size,
    T* result,
    const T* vector1,
    const typename GetTypeReal<T>::type* vector2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        result[i] = vector1[i] * vector2[i];
    }
}

template <typename T>
__global__ void vector_div_vector_kernel(
    const int size,
    T* result,
    const T* vector1,
    const typename GetTypeReal<T>::type* vector2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        result[i] = vector1[i] / vector2[i];
    }
}

template <typename T, typename Real>
__global__ void constantvector_addORsub_constantVector_kernel(
    const int size,
    T* result,
    const T* vector1,
    const Real constant1,
    const T* vector2,
    const Real constant2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        result[i] = vector1[i] * constant1 + vector2[i] * constant2;
    }
}

template <typename T>
__global__ void matrix_transpose_kernel(
        const int row,
        const int col,
    const T* in,
    T* out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < row)
    {
        for (int j = 0; j < col; j++)
        {
            out[j * row + i] = in[i * col + j];
        }
    }
}


template <typename T>
__global__ void matrix_setTo_another_kernel(
        const int n,
        const int LDA,
        const int LDB,
    const T* matrix_A,
    T* matrix_B)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < LDA && j < LDB)
    {
        for (int i = 0; i < n; i++)
        {
            matrix_B[i * LDB + j] = matrix_A[i * LDA + j];
        }
    }
}

template <typename T>
void line_minimize_with_block_op<T, base_device::DEVICE_GPU>::operator()(T* grad_out,
                                                                         T* hgrad_out,
                                                                         T* psi_out,
                                                                         T* hpsi_out,
                                                                         const int& n_basis,
                                                                         const int& n_basis_max,
                                                                         const int& n_band)
{
    auto A = reinterpret_cast<thrust::complex<Real>*>(grad_out);
    auto B = reinterpret_cast<thrust::complex<Real>*>(hgrad_out);
    auto C = reinterpret_cast<thrust::complex<Real>*>(psi_out);
    auto D = reinterpret_cast<thrust::complex<Real>*>(hpsi_out);

    line_minimize_with_block<Real><<<n_band, thread_per_block>>>(
            A, B, C, D,
            n_basis, n_basis_max);

    cudaCheckOnDebug();
}

template <typename T>
void calc_grad_with_block_op<T, base_device::DEVICE_GPU>::operator()(const Real* prec_in,
                                                                     Real* err_out,
                                                                     Real* beta_out,
                                                                     T* psi_out,
                                                                     T* hpsi_out,
                                                                     T* grad_out,
                                                                     T* grad_old_out,
                                                                     const int& n_basis,
                                                                     const int& n_basis_max,
                                                                     const int& n_band)
{
    auto A = reinterpret_cast<thrust::complex<Real>*>(psi_out);
    auto B = reinterpret_cast<thrust::complex<Real>*>(hpsi_out);
    auto C = reinterpret_cast<thrust::complex<Real>*>(grad_out);
    auto D = reinterpret_cast<thrust::complex<Real>*>(grad_old_out);

    calc_grad_with_block<Real><<<n_band, thread_per_block>>>(
            prec_in, err_out, beta_out,
            A, B, C, D,
            n_basis, n_basis_max);

    cudaCheckOnDebug();
}

template <>
double dot_real_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                const int& dim,
                                                                const double* psi_L,
                                                                const double* psi_R,
                                                                const bool reduce)
{
    double result = 0.0;
    xdot_wrapper(dim, psi_L, 1, psi_R, 1, result);
    if (reduce) {
        Parallel_Reduce::reduce_pool(result);
    }
    return result;
}
// for this implementation, please check
// https://thrust.github.io/doc/group__transformed__reductions_ga321192d85c5f510e52300ae762c7e995.html denghui modify
// 2022-10-03 Note that ddot_(2*dim,a,1,b,1) = REAL( zdotc_(dim,a,1,b,1) ) GPU specialization of actual computation.
template <typename FPTYPE>
inline FPTYPE dot_complex_wrapper(const base_device::DEVICE_GPU* d,
                                  const int& dim,
                                  const std::complex<FPTYPE>* psi_L,
                                  const std::complex<FPTYPE>* psi_R,
                                  const bool reduce)
{
    //<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
    // denghui modify 2022-10-07
    // Note that  ddot_(2*dim,a,1,b,1) = REAL( zdotc_(dim,a,1,b,1) )
    const FPTYPE* pL = reinterpret_cast<const FPTYPE*>(psi_L);
    const FPTYPE* pR = reinterpret_cast<const FPTYPE*>(psi_R);
    FPTYPE result = 0.0;
    xdot_wrapper(dim * 2, pL, 1, pR, 1, result);
    if (reduce) {
        Parallel_Reduce::reduce_pool(result);
    }
    return result;
}

template <>
float dot_real_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                            const int& dim,
                                                                            const std::complex<float>* psi_L,
                                                                            const std::complex<float>* psi_R,
                                                                            const bool reduce)
{
    return dot_complex_wrapper(d, dim, psi_L, psi_R, reduce);
}
template <>
double dot_real_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                              const int& dim,
                                                                              const std::complex<double>* psi_L,
                                                                              const std::complex<double>* psi_R,
                                                                              const bool reduce)
{
    return dot_complex_wrapper(d, dim, psi_L, psi_R, reduce);
}

// vector operator: result[i] = vector[i] / constant
template <>
void vector_div_constant_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                         const int dim,
                                                                         double* result,
                                                                         const double* vector,
                                                                         const double constant)
{
    // In small cases, 1024 threads per block will only utilize 17 blocks, much less than 40
    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    vector_div_constant_kernel<double> <<<block, thread >>> (dim, result, vector, constant);

    cudaCheckOnDebug();
}

// vector operator: result[i] = vector[i] / constant
template <typename FPTYPE>
inline void vector_div_constant_complex_wrapper(const base_device::DEVICE_GPU* d,
                                                const int dim,
                                                std::complex<FPTYPE>* result,
                                                const std::complex<FPTYPE>* vector,
                                                const FPTYPE constant)
{
    thrust::complex<FPTYPE>* result_tmp = reinterpret_cast<thrust::complex<FPTYPE>*>(result);
    const thrust::complex<FPTYPE>* vector_tmp = reinterpret_cast<const thrust::complex<FPTYPE>*>(vector);

    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    vector_div_constant_kernel<thrust::complex<FPTYPE>> <<<block, thread >>> (dim, result_tmp, vector_tmp, constant);

    cudaCheckOnDebug();
}
template <>
void vector_div_constant_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                                      const int dim,
                                                                                      std::complex<float>* result,
                                                                                      const std::complex<float>* vector,
                                                                                      const float constant)
{
    vector_div_constant_complex_wrapper(d, dim, result, vector, constant);
}
template <>
void vector_div_constant_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* d,
    const int dim,
    std::complex<double>* result,
    const std::complex<double>* vector,
    const double constant)
{
    vector_div_constant_complex_wrapper(d, dim, result, vector, constant);
}
// vector operator: result[i] = vector1[i](not complex) * vector2[i](not complex)
template <>
void vector_mul_vector_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                       const int& dim,
                                                                       double* result,
                                                                       const double* vector1,
                                                                       const double* vector2)
{
    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    vector_mul_vector_kernel<double> <<<block, thread >>> (dim, result, vector1, vector2);

    cudaCheckOnDebug();
}
// vector operator: result[i] = vector1[i](complex) * vector2[i](not complex)
template <typename FPTYPE>
inline void vector_mul_vector_complex_wrapper(const base_device::DEVICE_GPU* d,
                                              const int& dim,
                                              std::complex<FPTYPE>* result,
                                              const std::complex<FPTYPE>* vector1,
                                              const FPTYPE* vector2)
{
    thrust::complex<FPTYPE>* result_tmp = reinterpret_cast<thrust::complex<FPTYPE>*>(result);
    const thrust::complex<FPTYPE>* vector1_tmp = reinterpret_cast<const thrust::complex<FPTYPE>*>(vector1);
    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    vector_mul_vector_kernel<thrust::complex<FPTYPE>> <<<block, thread >>> (dim, result_tmp, vector1_tmp, vector2);

    cudaCheckOnDebug();
}
template <>
void vector_mul_vector_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                                    const int& dim,
                                                                                    std::complex<float>* result,
                                                                                    const std::complex<float>* vector1,
                                                                                    const float* vector2)
{
    vector_mul_vector_complex_wrapper(d, dim, result, vector1, vector2);
}
template <>
void vector_mul_vector_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* d,
    const int& dim,
    std::complex<double>* result,
    const std::complex<double>* vector1,
    const double* vector2)
{
    vector_mul_vector_complex_wrapper(d, dim, result, vector1, vector2);
}

// vector operator: result[i] = vector1[i](not complex) / vector2[i](not complex)
template <>
void vector_div_vector_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                       const int& dim,
                                                                       double* result,
                                                                       const double* vector1,
                                                                       const double* vector2)
{
    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    vector_div_vector_kernel<double> <<<block, thread >>> (dim, result, vector1, vector2);

    cudaCheckOnDebug();
}
// vector operator: result[i] = vector1[i](complex) / vector2[i](not complex)
template <typename FPTYPE>
inline void vector_div_vector_complex_wrapper(const base_device::DEVICE_GPU* d,
                                              const int& dim,
                                              std::complex<FPTYPE>* result,
                                              const std::complex<FPTYPE>* vector1,
                                              const FPTYPE* vector2)
{
    thrust::complex<FPTYPE>* result_tmp = reinterpret_cast<thrust::complex<FPTYPE>*>(result);
    const thrust::complex<FPTYPE>* vector1_tmp = reinterpret_cast<const thrust::complex<FPTYPE>*>(vector1);
    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    vector_div_vector_kernel<thrust::complex<FPTYPE>> <<<block, thread >>> (dim, result_tmp, vector1_tmp, vector2);

    cudaCheckOnDebug();
}
template <>
void vector_div_vector_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                                    const int& dim,
                                                                                    std::complex<float>* result,
                                                                                    const std::complex<float>* vector1,
                                                                                    const float* vector2)
{
    vector_div_vector_complex_wrapper(d, dim, result, vector1, vector2);
}
template <>
void vector_div_vector_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* d,
    const int& dim,
    std::complex<double>* result,
    const std::complex<double>* vector1,
    const double* vector2)
{
    vector_div_vector_complex_wrapper(d, dim, result, vector1, vector2);
}
// vector operator: result[i] = vector1[i] * constant1 + vector2[i] * constant2
template <typename T>
void constantvector_addORsub_constantVector_op<T, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                                       const int& dim,
                                                                                       T* result,
                                                                                       const T* vector1,
                                                                                       const Real constant1,
                                                                                       const T* vector2,
                                                                                       const Real constant2)
{
    using Type = typename GetTypeThrust<T>::type;
    using Real = typename GetTypeReal<T>::type;

    auto result_tmp = reinterpret_cast<Type*>(result);
    auto vector1_tmp = reinterpret_cast<const Type*>(vector1);
    auto vector2_tmp = reinterpret_cast<const Type*>(vector2);

    int thread = thread_per_block;
    int block = (dim + thread - 1) / thread;
    constantvector_addORsub_constantVector_kernel<Type, Real> <<<block, thread >>>(dim, result_tmp, vector1_tmp, constant1, vector2_tmp, constant2);

    cudaCheckOnDebug();
}

template <>
void axpy_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                          const int& N,
                                                          const double* alpha,
                                                          const double* X,
                                                          const int& incX,
                                                          double* Y,
                                                          const int& incY)
{
    cublasErrcheck(hipblasDaxpy(cublas_handle, N, alpha, X, incX, Y, incY));
}

template <>
void axpy_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                       const int& N,
                                                                       const std::complex<float>* alpha,
                                                                       const std::complex<float>* X,
                                                                       const int& incX,
                                                                       std::complex<float>* Y,
                                                                       const int& incY)
{
    cublasErrcheck(hipblasCaxpy(cublas_handle, N, (float2*)alpha, (float2*)X, incX, (float2*)Y, incY));
}

template <>
void axpy_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                        const int& N,
                                                                        const std::complex<double>* alpha,
                                                                        const std::complex<double>* X,
                                                                        const int& incX,
                                                                        std::complex<double>* Y,
                                                                        const int& incY)
{
    cublasErrcheck(hipblasZaxpy(cublas_handle, N, (double2*)alpha, (double2*)X, incX, (double2*)Y, incY));
}

hipblasOperation_t judge_trans_op(bool is_complex, const char& trans, const char* name)
{
    if (trans == 'N')
    {
        return HIPBLAS_OP_N;
    }
    else if(trans == 'T')
    {
        return HIPBLAS_OP_T;
    }
    else if(is_complex && trans == 'C')
    {
        return HIPBLAS_OP_C;
    }
    else 
    {
        ModuleBase::WARNING_QUIT(name, std::string("Unknown trans type ") + trans + std::string(" !"));
    }
}

template <>
void gemv_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                          const char& trans,
                                                          const int& m,
                                                          const int& n,
                                                          const double* alpha,
                                                          const double* A,
                                                          const int& lda,
                                                          const double* X,
                                                          const int& incx,
                                                          const double* beta,
                                                          double* Y,
                                                          const int& incy)
{
    hipblasOperation_t cutrans = judge_trans_op(false, trans, "gemv_op");
    cublasErrcheck(hipblasDgemv(cublas_handle, cutrans, m, n, alpha, A, lda, X, incx, beta, Y, incx));
}

template <>
void gemv_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                       const char& trans,
                                                                       const int& m,
                                                                       const int& n,
                                                                       const std::complex<float>* alpha,
                                                                       const std::complex<float>* A,
                                                                       const int& lda,
                                                                       const std::complex<float>* X,
                                                                       const int& incx,
                                                                       const std::complex<float>* beta,
                                                                       std::complex<float>* Y,
                                                                       const int& incy)
{
    hipblasOperation_t cutrans = judge_trans_op(true, trans, "gemv_op");
    cublasErrcheck(hipblasCgemv(cublas_handle, cutrans, m, n, (float2*)alpha, (float2*)A, lda, (float2*)X, incx, (float2*)beta, (float2*)Y, incx));
}

template <>
void gemv_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                        const char& trans,
                                                                        const int& m,
                                                                        const int& n,
                                                                        const std::complex<double>* alpha,
                                                                        const std::complex<double>* A,
                                                                        const int& lda,
                                                                        const std::complex<double>* X,
                                                                        const int& incx,
                                                                        const std::complex<double>* beta,
                                                                        std::complex<double>* Y,
                                                                        const int& incy)
{
    hipblasOperation_t cutrans = judge_trans_op(true, trans, "gemv_op");
    cublasErrcheck(hipblasZgemv(cublas_handle, cutrans, m, n, (double2*)alpha, (double2*)A, lda, (double2*)X, incx, (double2*)beta, (double2*)Y, incx));
}

template <>
void scal_op<float, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                         const int& N,
                                                         const std::complex<float>* alpha,
                                                         std::complex<float>* X,
                                                         const int& incx)
{
    cublasErrcheck(hipblasCscal(cublas_handle, N, (float2*)alpha, (float2*)X, incx));
}

template <>
void scal_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                          const int& N,
                                                          const std::complex<double>* alpha,
                                                          std::complex<double>* X,
                                                          const int& incx)
{
    cublasErrcheck(hipblasZscal(cublas_handle, N, (double2*)alpha, (double2*)X, incx));
}

template <>
void gemm_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                          const char& transa,
                                                          const char& transb,
                                                          const int& m,
                                                          const int& n,
                                                          const int& k,
                                                          const double* alpha,
                                                          const double* a,
                                                          const int& lda,
                                                          const double* b,
                                                          const int& ldb,
                                                          const double* beta,
                                                          double* c,
                                                          const int& ldc)
{
    hipblasOperation_t cutransA = judge_trans_op(false, transa, "gemm_op");
    hipblasOperation_t cutransB = judge_trans_op(false, transb, "gemm_op");
    cublasErrcheck(hipblasDgemm(cublas_handle, cutransA, cutransB, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc));
}
template <>
void gemm_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                       const char& transa,
                                                                       const char& transb,
                                                                       const int& m,
                                                                       const int& n,
                                                                       const int& k,
                                                                       const std::complex<float>* alpha,
                                                                       const std::complex<float>* a,
                                                                       const int& lda,
                                                                       const std::complex<float>* b,
                                                                       const int& ldb,
                                                                       const std::complex<float>* beta,
                                                                       std::complex<float>* c,
                                                                       const int& ldc)
{
    hipblasOperation_t cutransA = judge_trans_op(true, transa, "gemm_op");
    hipblasOperation_t cutransB = judge_trans_op(true, transb, "gemm_op");
    cublasErrcheck(hipblasCgemm(cublas_handle, cutransA, cutransB, m, n ,k, (float2*)alpha, (float2*)a , lda, (float2*)b, ldb, (float2*)beta, (float2*)c, ldc));
}

template <>
void gemm_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                        const char& transa,
                                                                        const char& transb,
                                                                        const int& m,
                                                                        const int& n,
                                                                        const int& k,
                                                                        const std::complex<double>* alpha,
                                                                        const std::complex<double>* a,
                                                                        const int& lda,
                                                                        const std::complex<double>* b,
                                                                        const int& ldb,
                                                                        const std::complex<double>* beta,
                                                                        std::complex<double>* c,
                                                                        const int& ldc)
{
    hipblasOperation_t cutransA = judge_trans_op(true, transa, "gemm_op");
    hipblasOperation_t cutransB = judge_trans_op(true, transb, "gemm_op");
    cublasErrcheck(hipblasZgemm(cublas_handle, cutransA, cutransB, m, n ,k, (double2*)alpha, (double2*)a , lda, (double2*)b, ldb, (double2*)beta, (double2*)c, ldc));
}

template <>
void matrixTranspose_op<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                     const int& row,
                                                                     const int& col,
                                                                     const double* input_matrix,
                                                                     double* output_matrix)
{
    double* device_temp = nullptr;
    base_device::memory::resize_memory_op<double, base_device::DEVICE_GPU>()(d, device_temp, row * col);

    if (row == col)
    {
        double ONE = 1.0, ZERO = 0.0;

        // use 'geam' API todo transpose.
        cublasErrcheck(hipblasDgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, col, row, &ONE, input_matrix, col, &ZERO, input_matrix, col, device_temp, col));
    }
    else
    {
        int thread = 1024;
        int block = (row + col + thread - 1) / thread;
        matrix_transpose_kernel<double> <<<block, thread >>> (row, col, input_matrix, device_temp);

        cudaCheckOnDebug();
    }

    base_device::memory::synchronize_memory_op<double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>()(
        d,
        d,
        output_matrix,
        device_temp,
        row * col);

    base_device::memory::delete_memory_op<double, base_device::DEVICE_GPU>()(d, device_temp);
}

template <>
void matrixTranspose_op<std::complex<float>, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* d,
    const int& row,
    const int& col,
    const std::complex<float>* input_matrix,
    std::complex<float>* output_matrix)
{
    std::complex<float>* device_temp = nullptr;
    base_device::memory::resize_memory_op<std::complex<float>, base_device::DEVICE_GPU>()(d, device_temp, row * col);

    if (row == col)
    {
        double2 ONE, ZERO;
        ONE.x = 1.0;
        ONE.y = 0.0;
        ZERO.x = ZERO.y = 0.0;

        // use 'geam' API todo transpose.
        cublasErrcheck(hipblasCgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, col, row,
                                   reinterpret_cast<const float2 *>(&ONE), (float2*)input_matrix, col,
                                   reinterpret_cast<const float2 *>(&ZERO), (float2*)input_matrix, col, (float2*)device_temp, col));
    } else
    {
        int thread = 1024;
        int block = (row + col + thread - 1) / thread;
        matrix_transpose_kernel<thrust::complex<float>> <<<block, thread >>> (row, col, (thrust::complex<float>*)input_matrix, (thrust::complex<float>*)device_temp);

        cudaCheckOnDebug();
    }

    base_device::memory::synchronize_memory_op<std::complex<float>, base_device::DEVICE_GPU, base_device::DEVICE_GPU>()(
        d,
        d,
        output_matrix,
        device_temp,
        row * col);

    base_device::memory::delete_memory_op<std::complex<float>, base_device::DEVICE_GPU>()(d, device_temp);

    cudaCheckOnDebug();

}

template <>
void matrixTranspose_op<std::complex<double>, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* d,
    const int& row,
    const int& col,
    const std::complex<double>* input_matrix,
    std::complex<double>* output_matrix)
{
    std::complex<double>* device_temp = nullptr;
    base_device::memory::resize_memory_op<std::complex<double>, base_device::DEVICE_GPU>()(d, device_temp, row * col);

    if (row == col)
    {
        double2 ONE, ZERO;
        ONE.x = 1.0;
        ONE.y = 0.0;
        ZERO.x = ZERO.y = 0.0;

        // use 'geam' API todo transpose.
        cublasErrcheck(hipblasZgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, col, row, &ONE, (double2*)input_matrix, col, &ZERO, (double2*)input_matrix, col, (double2*)device_temp, col));
    } else
    {
        int thread = 1024;
        int block = (row + col + thread - 1) / thread;
        matrix_transpose_kernel<thrust::complex<double>> <<<block, thread >>> (row, col, (thrust::complex<double>*)input_matrix, (thrust::complex<double>*)device_temp);
        cudaCheckOnDebug();
    }

    base_device::memory::synchronize_memory_op<std::complex<double>,
                                               base_device::DEVICE_GPU,
                                               base_device::DEVICE_GPU>()(d, d, output_matrix, device_temp, row * col);

    base_device::memory::delete_memory_op<std::complex<double>, base_device::DEVICE_GPU>()(d, device_temp);
}

template <>
void matrixSetToAnother<double, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                     const int& n,
                                                                     const double* A,
                                                                     const int& LDA,
                                                                     double* B,
                                                                     const int& LDB)
{
    int thread = 1024;
    int block = (LDA + thread - 1) / thread;
    matrix_setTo_another_kernel<double> <<<block, thread >>> (n, LDA, LDB, A, B);
    cudaCheckOnDebug();
}
template <>
void matrixSetToAnother<std::complex<float>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                                  const int& n,
                                                                                  const std::complex<float>* A,
                                                                                  const int& LDA,
                                                                                  std::complex<float>* B,
                                                                                  const int& LDB)
{
    int thread = 1024;
    int block = (LDA + thread - 1) / thread;
    matrix_setTo_another_kernel<thrust::complex<float>> <<<block, thread >>> (n, LDA, LDB, reinterpret_cast<const thrust::complex<float>*>(A), reinterpret_cast<thrust::complex<float>*>(B));
    cudaCheckOnDebug();
}
template <>
void matrixSetToAnother<std::complex<double>, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* d,
                                                                                   const int& n,
                                                                                   const std::complex<double>* A,
                                                                                   const int& LDA,
                                                                                   std::complex<double>* B,
                                                                                   const int& LDB)
{
    int thread = 1024;
    int block = (LDA + thread - 1) / thread;
    matrix_setTo_another_kernel<thrust::complex<double>> <<<block, thread >>> (n, LDA, LDB, reinterpret_cast<const thrust::complex<double>*>(A), reinterpret_cast<thrust::complex<double>*>(B));

    cudaCheckOnDebug();
}


// Explicitly instantiate functors for the types of functor registered.
template struct dot_real_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct calc_grad_with_block_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct line_minimize_with_block_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct vector_div_constant_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct vector_mul_vector_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct vector_div_vector_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct constantvector_addORsub_constantVector_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct matrixSetToAnother<std::complex<float>, base_device::DEVICE_GPU>;

template struct dot_real_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct calc_grad_with_block_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct line_minimize_with_block_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct vector_div_constant_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct vector_mul_vector_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct vector_div_vector_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct constantvector_addORsub_constantVector_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct matrixSetToAnother<std::complex<double>, base_device::DEVICE_GPU>;

#ifdef __LCAO
template struct dot_real_op<double, base_device::DEVICE_GPU>;
template struct vector_div_constant_op<double, base_device::DEVICE_GPU>;
template struct vector_mul_vector_op<double, base_device::DEVICE_GPU>;
template struct vector_div_vector_op<double, base_device::DEVICE_GPU>;
template struct matrixSetToAnother<double, base_device::DEVICE_GPU>;
template struct constantvector_addORsub_constantVector_op<double, base_device::DEVICE_GPU>;
#endif
}  // namespace hsolver
