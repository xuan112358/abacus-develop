#include "kernels/cuda/cuda_tools.cuh"
#include "module_base/ylm.h"
#include "gint_rho_gpu.h"
#include "gint_tools.h"
#include "kernels/cuda/gint_rho.cuh"

#ifdef _OPENMP
#include <omp.h>
#endif

namespace GintKernel
{

void gint_rho_gpu(const hamilt::HContainer<double>* dm,
                        const double* ylmcoef_now,
                        const double dr,
                        const double* rcut,
                        const Grid_Technique& gridt,
                        const UnitCell& ucell,
                        double* rho)
{
    checkCuda(hipSetDevice(gridt.dev_id));
    // checkCuda(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

    const int nbzp = gridt.nbzp;
    const int nczp =nbzp * gridt.bz;
    const int num_mcell_on_proc = nczp * gridt.ncx * gridt.ncy;
    const int lgd = gridt.lgd;
    const int max_atom = gridt.max_atom;
    const int num_streams = gridt.nstreams;
    const int max_atom_per_bcell = max_atom * gridt.bxyz;
    const int max_atom_per_z = max_atom * nbzp;
    const int max_phi_per_z = max_atom_per_bcell * nbzp * ucell.nwmax;
    const int max_atompair_per_z = max_atom * max_atom * nbzp;

    std::vector<hipStream_t> streams(num_streams);
    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamCreate(&streams[i]));
    }

    Cuda_Mem_Wrapper<double> dr_part(max_atom_per_z * 3, num_streams, true);
    Cuda_Mem_Wrapper<uint8_t> atoms_type(max_atom_per_z, num_streams, true);
    // The first number in every group of two represents the number of atoms on that bigcell.
    // The second number represents the cumulative number of atoms up to that bigcell.
    Cuda_Mem_Wrapper<int> atoms_num_info(2 * nbzp, num_streams, true);

    Cuda_Mem_Wrapper<double> psi(max_phi_per_z, num_streams, false);
    Cuda_Mem_Wrapper<double> psi_dm(max_phi_per_z, num_streams, false);

    Cuda_Mem_Wrapper<double> gemm_alpha(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_m(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_n(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_k(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_lda(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldb(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<int> gemm_ldc(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_A(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_B(max_atompair_per_z, num_streams, true);
    Cuda_Mem_Wrapper<double*> gemm_C(max_atompair_per_z, num_streams, true);
    
    Cuda_Mem_Wrapper<double> rho_g(num_mcell_on_proc, 1, false);
    Cuda_Mem_Wrapper<double*> dot_product(nbzp * gridt.bxyz, num_streams, true);

    Cuda_Mem_Wrapper<double> dm_matrix(dm->get_nnr(), 1, false);
    // retrieve the density matrix on the host
    checkCuda(hipMemcpy(dm_matrix.get_device_pointer(),
                         dm->get_wrapper(),
                         dm->get_nnr() * sizeof(double),
                         hipMemcpyHostToDevice));

// calculate the rho for every nbzp bigcells
#ifdef _OPENMP
#pragma omp parallel for num_threads(num_streams) collapse(2)
#endif
    for (int i = 0; i < gridt.nbx; i++)
    {
        for (int j = 0; j < gridt.nby; j++)
        {
            // 20240620 Note that it must be set again here because 
            // cuda's device is not safe in a multi-threaded environment.

            checkCuda(hipSetDevice(gridt.dev_id));
            // get stream id
#ifdef _OPENMP
            const int sid = omp_get_thread_num();
#else
            const int sid = 0;
#endif

            int max_m = 0;
            int max_n = 0;
            int atom_pair_num = 0;
            int atoms_per_z = 0;
            const int grid_index_ij = i * gridt.nby * nbzp + j * nbzp;

            // generate GPU tasks, including the calculation of psir, matrix
            // multiplication, and dot product
            gtask_rho(gridt,
                      grid_index_ij,
                      ucell,
                      dr_part.get_host_pointer(sid),
                      atoms_type.get_host_pointer(sid),
                      atoms_num_info.get_host_pointer(sid),
                      atoms_per_z);

            alloc_mult_dot_rho(
                dm,
                gridt,
                ucell,
                grid_index_ij,
                max_atom,
                lgd,
                nczp,
                atoms_num_info.get_host_pointer(sid),
                psi.get_device_pointer(sid),
                psi_dm.get_device_pointer(sid),
                dm_matrix.get_device_pointer(),
                gemm_alpha.get_host_pointer(sid),
                gemm_m.get_host_pointer(sid),
                gemm_n.get_host_pointer(sid),
                gemm_k.get_host_pointer(sid),
                gemm_lda.get_host_pointer(sid),
                gemm_ldb.get_host_pointer(sid),
                gemm_ldc.get_host_pointer(sid),
                gemm_A.get_host_pointer(sid),
                gemm_B.get_host_pointer(sid),
                gemm_C.get_host_pointer(sid),
                max_m,
                max_n,
                atom_pair_num,
                rho_g.get_device_pointer(),
                dot_product.get_host_pointer(sid));
            
            dr_part.copy_host_to_device_async(streams[sid], sid, atoms_per_z * 3);
            atoms_type.copy_host_to_device_async(streams[sid], sid, atoms_per_z);
            atoms_num_info.copy_host_to_device_async(streams[sid], sid);

            gemm_alpha.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_m.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_n.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_k.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_lda.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldb.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_ldc.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_A.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_B.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            gemm_C.copy_host_to_device_async(streams[sid], sid, atom_pair_num);
            dot_product.copy_host_to_device_async(streams[sid], sid);
            
            psi.memset_device_async(streams[sid], sid, 0);
            psi_dm.memset_device_async(streams[sid], sid, 0);

            // Launching kernel to calculate psi
            dim3 grid_psi(nbzp, gridt.bxyz);
            dim3 block_psi(64);
            get_psi<<<grid_psi, block_psi, 0, streams[sid]>>>(
                gridt.ylmcoef_g,
                dr,
                gridt.bxyz,
                ucell.nwmax,
                max_atom,
                gridt.atom_nwl_g,
                gridt.atom_new_g,
                gridt.atom_ylm_g,
                gridt.atom_nw_g,
                gridt.rcut_g,
                gridt.nr_max,
                gridt.psi_u_g,
                gridt.mcell_pos_g,
                dr_part.get_device_pointer(sid),
                atoms_type.get_device_pointer(sid),
                atoms_num_info.get_device_pointer(sid),
                psi.get_device_pointer(sid));
            checkCudaLastError();

            // Performing matrix multiplication alpha * mat_dm * mat_psir
            gridt.fastest_matrix_mul(max_m,
                                     max_n,
                                     gemm_m.get_device_pointer(sid),
                                     gemm_n.get_device_pointer(sid),
                                     gemm_k.get_device_pointer(sid),
                                     gemm_A.get_device_pointer(sid),
                                     gemm_lda.get_device_pointer(sid),
                                     gemm_B.get_device_pointer(sid),
                                     gemm_ldb.get_device_pointer(sid),
                                     gemm_C.get_device_pointer(sid),
                                     gemm_ldc.get_device_pointer(sid),
                                     atom_pair_num,
                                     streams[sid],
                                     gemm_alpha.get_device_pointer(sid));
            checkCudaLastError();

            // Launching kernel to calculate dot product psir * psir_dm
            // if warpSize is not eauql to 32, the psir_dot kernel should be modified
            dim3 grid_dot(nbzp, gridt.bxyz);
            dim3 block_dot(64); 
            psir_dot<<<grid_dot, block_dot, sizeof(double) * 32, streams[sid]>>>(
                gridt.bxyz,
                ucell.nwmax,
                atoms_num_info.get_device_pointer(sid),
                psi.get_device_pointer(sid),
                psi_dm.get_device_pointer(sid),
                dot_product.get_device_pointer(sid));
            checkCudaLastError();
            checkCuda(hipStreamSynchronize(streams[sid]));
        }
    }

    // Copy rho from device to host
    checkCuda(hipMemcpy(rho,
                         rho_g.get_device_pointer(),
                         num_mcell_on_proc * sizeof(double),
                         hipMemcpyDeviceToHost));

    for (int i = 0; i < num_streams; i++)
    {
        checkCuda(hipStreamDestroy(streams[i]));
    }
}
} // namespace GintKernel
