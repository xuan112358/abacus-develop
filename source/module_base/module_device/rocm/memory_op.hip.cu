#include "hip/hip_runtime.h"
#include "module_base/module_device/memory_op.h"

#include <base/macros/macros.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include <complex>
#include <type_traits>

#define THREADS_PER_BLOCK 256

namespace base_device
{
namespace memory
{

template <typename FPTYPE_out, typename FPTYPE_in>
__global__ void cast_memory(FPTYPE_out* out, const FPTYPE_in* in, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
    {
        return;
    }
    out[idx] = static_cast<FPTYPE_out>(in[idx]);
}

template <typename FPTYPE_out, typename FPTYPE_in>
__global__ void cast_memory(std::complex<FPTYPE_out>* out, const std::complex<FPTYPE_in>* in, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
    {
        return;
    }
    auto* _out = reinterpret_cast<thrust::complex<FPTYPE_out>*>(out);
    const auto* _in = reinterpret_cast<const thrust::complex<FPTYPE_in>*>(in);
    _out[idx] = static_cast<thrust::complex<FPTYPE_out>>(_in[idx]);
}

template <typename FPTYPE>
void resize_memory_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                                   FPTYPE*& arr,
                                                                   const size_t size,
                                                                   const char* record_in)
{
    if (arr != nullptr)
    {
        delete_memory_op<FPTYPE, base_device::DEVICE_GPU>()(dev, arr);
    }
    hipErrcheck(hipMalloc((void**)&arr, sizeof(FPTYPE) * size));
}

template <typename FPTYPE>
void set_memory_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                                FPTYPE* arr,
                                                                const int var,
                                                                const size_t size)
{
    hipErrcheck(hipMemset(arr, var, sizeof(FPTYPE) * size));
}

template <typename FPTYPE>
void synchronize_memory_op<FPTYPE, base_device::DEVICE_CPU, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_CPU* dev_out,
    const base_device::DEVICE_GPU* dev_in,
    FPTYPE* arr_out,
    const FPTYPE* arr_in,
    const size_t size)
{
    hipErrcheck(hipMemcpy(arr_out, arr_in, sizeof(FPTYPE) * size, hipMemcpyDeviceToHost));
}

template <typename FPTYPE>
void synchronize_memory_op<FPTYPE, base_device::DEVICE_GPU, base_device::DEVICE_CPU>::operator()(
    const base_device::DEVICE_GPU* dev_out,
    const base_device::DEVICE_CPU* dev_in,
    FPTYPE* arr_out,
    const FPTYPE* arr_in,
    const size_t size)
{
    hipErrcheck(hipMemcpy(arr_out, arr_in, sizeof(FPTYPE) * size, hipMemcpyHostToDevice));
}

template <typename FPTYPE>
void synchronize_memory_op<FPTYPE, base_device::DEVICE_GPU, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* dev_out,
    const base_device::DEVICE_GPU* dev_in,
    FPTYPE* arr_out,
    const FPTYPE* arr_in,
    const size_t size)
{
    hipErrcheck(hipMemcpy(arr_out, arr_in, sizeof(FPTYPE) * size, hipMemcpyDeviceToDevice));
}

template <typename FPTYPE_out, typename FPTYPE_in>
struct cast_memory_op<FPTYPE_out, FPTYPE_in, base_device::DEVICE_GPU, base_device::DEVICE_GPU> {
    void operator()(const base_device::DEVICE_GPU* dev_out,
                    const base_device::DEVICE_GPU* dev_in,
                    FPTYPE_out* arr_out,
                    const FPTYPE_in* arr_in,
                    const size_t size) {

        if (size == 0) {return;}
        const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        hipLaunchKernelGGL(cast_memory, dim3(block), dim3(THREADS_PER_BLOCK), 0, 0, arr_out, arr_in, size);
        hipCheckOnDebug();
    }
};

template <typename FPTYPE_out, typename FPTYPE_in>
struct cast_memory_op<FPTYPE_out, FPTYPE_in, base_device::DEVICE_GPU, base_device::DEVICE_CPU> {
    void operator()(const base_device::DEVICE_GPU* dev_out,
                    const base_device::DEVICE_CPU* dev_in,
                    FPTYPE_out* arr_out,
                    const FPTYPE_in* arr_in,
                    const size_t size) {

        if (size == 0) {return;}
        // No need to cast the memory if the data types are the same.
        if (std::is_same<FPTYPE_out, FPTYPE_in>::value)
        {
            synchronize_memory_op<FPTYPE_out, base_device::DEVICE_GPU, base_device::DEVICE_CPU>()(dev_out,
                                                                                                  dev_in,
                                                                                                  arr_out,
                                                                                                  reinterpret_cast<const FPTYPE_out*>(arr_in),
                                                                                                  size);
            return;
        }
        FPTYPE_in * arr = nullptr;
        hipErrcheck(hipMalloc((void **)&arr, sizeof(FPTYPE_in) * size));
        hipErrcheck(hipMemcpy(arr, arr_in, sizeof(FPTYPE_in) * size, hipMemcpyHostToDevice));
        const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        hipLaunchKernelGGL(cast_memory, dim3(block), dim3(THREADS_PER_BLOCK), 0, 0, arr_out, arr, size);
        hipCheckOnDebug();
        hipErrcheck(hipFree(arr));
    }
};

template <typename FPTYPE_out, typename FPTYPE_in>
struct cast_memory_op<FPTYPE_out, FPTYPE_in, base_device::DEVICE_CPU, base_device::DEVICE_GPU> {
    void operator()(const base_device::DEVICE_CPU* dev_out,
                    const base_device::DEVICE_GPU* dev_in,
                    FPTYPE_out* arr_out,
                    const FPTYPE_in* arr_in,
                    const size_t size) {

        if (size == 0) {return;}
        // No need to cast the memory if the data types are the same.
        if (std::is_same<FPTYPE_out, FPTYPE_in>::value)
        {
            synchronize_memory_op<FPTYPE_out, base_device::DEVICE_CPU, base_device::DEVICE_GPU>()(dev_out,
                                                                                                  dev_in,
                                                                                                  arr_out,
                                                                                                  reinterpret_cast<const FPTYPE_out*>(arr_in),
                                                                                                  size);
            return;
        }
        auto * arr = (FPTYPE_in*) malloc(sizeof(FPTYPE_in) * size);
        hipErrcheck(hipMemcpy(arr, arr_in, sizeof(FPTYPE_in) * size, hipMemcpyDeviceToHost));
        for (int ii = 0; ii < size; ii++) {
            arr_out[ii] = static_cast<FPTYPE_out>(arr[ii]);
        }
        free(arr);
    }
};

template <typename FPTYPE>
void delete_memory_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev, FPTYPE* arr)
{
    hipErrcheck(hipFree(arr));
}

template struct resize_memory_op<int, base_device::DEVICE_GPU>;
template struct resize_memory_op<float, base_device::DEVICE_GPU>;
template struct resize_memory_op<double, base_device::DEVICE_GPU>;
template struct resize_memory_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct resize_memory_op<std::complex<double>, base_device::DEVICE_GPU>;

template struct set_memory_op<int, base_device::DEVICE_GPU>;
template struct set_memory_op<float, base_device::DEVICE_GPU>;
template struct set_memory_op<double, base_device::DEVICE_GPU>;
template struct set_memory_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct set_memory_op<std::complex<double>, base_device::DEVICE_GPU>;

template struct synchronize_memory_op<int, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<int, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<int, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<float, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<float, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<double, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<double, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<float>, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<float>, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<std::complex<float>, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<double>, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<double>, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<std::complex<double>, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;

template struct cast_memory_op<float, float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<float, double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>, float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>, double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<float, float, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<double, double, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<float, double, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<double, float, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<float, float, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, double, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<float, double, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, float, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<float>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<double>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<double>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<float>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;

template struct delete_memory_op<int, base_device::DEVICE_GPU>;
template struct delete_memory_op<float, base_device::DEVICE_GPU>;
template struct delete_memory_op<double, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct delete_memory_op<float*, base_device::DEVICE_GPU>;
template struct delete_memory_op<double*, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<float>*, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<double>*, base_device::DEVICE_GPU>;

} // namespace memory
} // end of namespace base_device