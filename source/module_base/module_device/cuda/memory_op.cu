#include "hip/hip_runtime.h"
#include "module_base/module_device/memory_op.h"
#include "module_base/memory.h"

#include <base/macros/macros.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

#include <complex>
#include <type_traits>

#define THREADS_PER_BLOCK 256

namespace base_device
{
namespace memory
{

template <typename FPTYPE_out, typename FPTYPE_in>
__global__ void cast_memory(FPTYPE_out* out, const FPTYPE_in* in, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
    {
        return;
    }
    out[idx] = static_cast<FPTYPE_out>(in[idx]);
}

template <typename FPTYPE_out, typename FPTYPE_in>
__global__ void cast_memory(std::complex<FPTYPE_out>* out, const std::complex<FPTYPE_in>* in, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
    {
        return;
    }
    auto* _out = reinterpret_cast<thrust::complex<FPTYPE_out>*>(out);
    const auto* _in = reinterpret_cast<const thrust::complex<FPTYPE_in>*>(in);
    _out[idx] = static_cast<thrust::complex<FPTYPE_out>>(_in[idx]);
}

template <typename FPTYPE_out, typename FPTYPE_in>
__global__ void cast_memory(std::complex<FPTYPE_out>* out, const FPTYPE_in* in, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
    {
        return;
    }
    auto* _out = reinterpret_cast<thrust::complex<FPTYPE_out>*>(out);
    _out[idx] = static_cast<thrust::complex<FPTYPE_out>>(in[idx]);
}

template <typename FPTYPE>
void resize_memory_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                                   FPTYPE*& arr,
                                                                   const size_t size,
                                                                   const char* record_in)
{
    if (arr != nullptr)
    {
        delete_memory_op<FPTYPE, base_device::DEVICE_GPU>()(dev, arr);
    }
    cudaErrcheck(hipMalloc((void**)&arr, sizeof(FPTYPE) * size));
    std::string record_string;
    if (record_in != nullptr)
    {
        record_string = record_in;
    }
    else
    {
        record_string = "no_record";
    }

    if (record_string != "no_record")
    {
        ModuleBase::Memory::record_gpu(record_string, sizeof(FPTYPE) * size);
    }
}

template <typename FPTYPE>
void set_memory_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                                FPTYPE* arr,
                                                                const int var,
                                                                const size_t size)
{
    cudaErrcheck(hipMemset(arr, var, sizeof(FPTYPE) * size));
}

template <typename FPTYPE>
void synchronize_memory_op<FPTYPE, base_device::DEVICE_CPU, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_CPU* dev_out,
    const base_device::DEVICE_GPU* dev_in,
    FPTYPE* arr_out,
    const FPTYPE* arr_in,
    const size_t size)
{
    cudaErrcheck(hipMemcpy(arr_out, arr_in, sizeof(FPTYPE) * size, hipMemcpyDeviceToHost));
}

template <typename FPTYPE>
void synchronize_memory_op<FPTYPE, base_device::DEVICE_GPU, base_device::DEVICE_CPU>::operator()(
    const base_device::DEVICE_GPU* dev_out,
    const base_device::DEVICE_CPU* dev_in,
    FPTYPE* arr_out,
    const FPTYPE* arr_in,
    const size_t size)
{
    cudaErrcheck(hipMemcpy(arr_out, arr_in, sizeof(FPTYPE) * size, hipMemcpyHostToDevice));
}

template <typename FPTYPE>
void synchronize_memory_op<FPTYPE, base_device::DEVICE_GPU, base_device::DEVICE_GPU>::operator()(
    const base_device::DEVICE_GPU* dev_out,
    const base_device::DEVICE_GPU* dev_in,
    FPTYPE* arr_out,
    const FPTYPE* arr_in,
    const size_t size)
{
    cudaErrcheck(hipMemcpy(arr_out, arr_in, sizeof(FPTYPE) * size, hipMemcpyDeviceToDevice));
}

template <typename FPTYPE_out, typename FPTYPE_in>
struct cast_memory_op<FPTYPE_out, FPTYPE_in, base_device::DEVICE_GPU, base_device::DEVICE_GPU>
{
    void operator()(const base_device::DEVICE_GPU* dev_out,
                    const base_device::DEVICE_GPU* dev_in,
                    FPTYPE_out* arr_out,
                    const FPTYPE_in* arr_in,
                    const size_t size)
    {
        if (size == 0)
        {
            return;
        }
        const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        cast_memory<<<block, THREADS_PER_BLOCK>>>(arr_out, arr_in, size);

        cudaCheckOnDebug();
    }
};

template <typename FPTYPE_out, typename FPTYPE_in>
struct cast_memory_op<FPTYPE_out, FPTYPE_in, base_device::DEVICE_GPU, base_device::DEVICE_CPU> {
    void operator()(const base_device::DEVICE_GPU* dev_out,
                    const base_device::DEVICE_CPU* dev_in,
                    FPTYPE_out* arr_out,
                    const FPTYPE_in* arr_in,
                    const size_t size) {

        if (size == 0) {return;}
        // No need to cast the memory if the data types are the same.
        if (std::is_same<FPTYPE_out, FPTYPE_in>::value)
        {
            synchronize_memory_op<FPTYPE_out, base_device::DEVICE_GPU, base_device::DEVICE_CPU>()(dev_out,
                                                                                                  dev_in,
                                                                                                  arr_out,
                                                                                                  reinterpret_cast<const FPTYPE_out*>(arr_in),
                                                                                                  size);
            return;
        }
        FPTYPE_in * arr = nullptr;
        cudaErrcheck(hipMalloc((void **)&arr, sizeof(FPTYPE_in) * size));
        cudaErrcheck(hipMemcpy(arr, arr_in, sizeof(FPTYPE_in) * size, hipMemcpyHostToDevice));
        const int block = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        cast_memory<<<block, THREADS_PER_BLOCK>>>(arr_out, arr, size);
        cudaCheckOnDebug();
        cudaErrcheck(hipFree(arr));
    }
};

template <typename FPTYPE_out, typename FPTYPE_in>
struct cast_memory_op<FPTYPE_out, FPTYPE_in, base_device::DEVICE_CPU, base_device::DEVICE_GPU> {
    void operator()(const base_device::DEVICE_CPU* dev_out,
                    const base_device::DEVICE_GPU* dev_in,
                    FPTYPE_out* arr_out,
                    const FPTYPE_in* arr_in,
                    const size_t size) {
        if (size == 0) {return;}
        // No need to cast the memory if the data types are the same.
        if (std::is_same<FPTYPE_out, FPTYPE_in>::value)
        {
            synchronize_memory_op<FPTYPE_out, base_device::DEVICE_CPU, base_device::DEVICE_GPU>()(dev_out,
                                                                                                  dev_in,
                                                                                                  arr_out,
                                                                                                  reinterpret_cast<const FPTYPE_out*>(arr_in),
                                                                                                  size);
            return;
        }
        auto * arr = (FPTYPE_in*) malloc(sizeof(FPTYPE_in) * size);
        cudaErrcheck(hipMemcpy(arr, arr_in, sizeof(FPTYPE_in) * size, hipMemcpyDeviceToHost));
        for (int ii = 0; ii < size; ii++) {
            arr_out[ii] = static_cast<FPTYPE_out>(arr[ii]);
        }
        free(arr);
    }
};

template <typename FPTYPE>
void delete_memory_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev, FPTYPE* arr)
{
    cudaErrcheck(hipFree(arr));
}

template struct resize_memory_op<int, base_device::DEVICE_GPU>;
template struct resize_memory_op<float, base_device::DEVICE_GPU>;
template struct resize_memory_op<double, base_device::DEVICE_GPU>;
template struct resize_memory_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct resize_memory_op<std::complex<double>, base_device::DEVICE_GPU>;

template struct set_memory_op<int, base_device::DEVICE_GPU>;
template struct set_memory_op<float, base_device::DEVICE_GPU>;
template struct set_memory_op<double, base_device::DEVICE_GPU>;
template struct set_memory_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct set_memory_op<std::complex<double>, base_device::DEVICE_GPU>;

template struct synchronize_memory_op<int, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<int, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<int, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<float, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<float, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<double, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<double, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<float>, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<float>, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<std::complex<float>, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<double>, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct synchronize_memory_op<std::complex<double>, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct synchronize_memory_op<std::complex<double>, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;

template struct cast_memory_op<float, float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<float, double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>, float, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>, double, base_device::DEVICE_GPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<float, float, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<double, double, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<float, double, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<double, float, base_device::DEVICE_GPU, base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<double>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<float>,
                               base_device::DEVICE_GPU,
                               base_device::DEVICE_CPU>;
template struct cast_memory_op<float, float, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, double, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<float, double, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<double, float, base_device::DEVICE_CPU, base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<float>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<double>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<float>,
                               std::complex<double>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;
template struct cast_memory_op<std::complex<double>,
                               std::complex<float>,
                               base_device::DEVICE_CPU,
                               base_device::DEVICE_GPU>;

template struct delete_memory_op<int, base_device::DEVICE_GPU>;
template struct delete_memory_op<float, base_device::DEVICE_GPU>;
template struct delete_memory_op<double, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<double>, base_device::DEVICE_GPU>;
template struct delete_memory_op<float*, base_device::DEVICE_GPU>;
template struct delete_memory_op<double*, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<float>*, base_device::DEVICE_GPU>;
template struct delete_memory_op<std::complex<double>*, base_device::DEVICE_GPU>;

} // namespace memory
} // end of namespace base_device