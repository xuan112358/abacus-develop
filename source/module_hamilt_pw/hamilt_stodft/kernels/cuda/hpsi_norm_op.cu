#include "hip/hip_runtime.h"
#include "module_hamilt_pw/hamilt_stodft/kernels/hpsi_norm_op.h"

#include <base/macros/macros.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>

namespace hamilt
{
#define THREADS_PER_BLOCK 256

template <typename FPTYPE>
__global__ void hpsi_norm(const int npwk_max,
                          const int npwk,
                          const FPTYPE Ebar,
                          const FPTYPE DeltaE,
                          thrust::complex<FPTYPE>* hpsi,
                          const thrust::complex<FPTYPE>* psi_in)
{
    const int block_idx = blockIdx.x;
    const int thread_idx = threadIdx.x;
    const int start_idx = block_idx * npwk_max;
    for (int ii = thread_idx; ii < npwk; ii += blockDim.x)
    {
        hpsi[start_idx + ii] = (hpsi[start_idx + ii] - Ebar * psi_in[start_idx + ii]) / DeltaE;
    }
}

template <typename FPTYPE>
void hamilt::hpsi_norm_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* dev,
                                                                       const int& nbands,
                                                                       const int& npwk_max,
                                                                       const int& npwk,
                                                                       const FPTYPE& Ebar,
                                                                       const FPTYPE& DeltaE,
                                                                       std::complex<FPTYPE>* hpsi,
                                                                       const std::complex<FPTYPE>* psi_in)
{
    // <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
    hpsi_norm<FPTYPE><<<nbands, THREADS_PER_BLOCK>>>(
      npwk_max, npwk, Ebar, DeltaE,
      reinterpret_cast<thrust::complex<FPTYPE>*>(hpsi),
      reinterpret_cast<const thrust::complex<FPTYPE>*>(psi_in));
    cudaCheckOnDebug();
}

template struct hpsi_norm_op<float, base_device::DEVICE_GPU>;
template struct hpsi_norm_op<double, base_device::DEVICE_GPU>;

} // namespace hamilt