#include "hip/hip_runtime.h"
#include "module_hamilt_pw/hamilt_pwdft/kernels/stress_op.h"
#include "vnl_tools_cu.hpp"
#include "module_base/module_device/types.h"

#include <complex>
#include <thrust/complex.h>
#include <base/macros/macros.h>
#include <module_base/module_device/device.h>

#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define FULL_MASK 0xffffffff
#define WARP_SIZE 32

namespace hamilt{

template <typename FPTYPE>
__forceinline__
__device__
void warp_reduce(FPTYPE & val) {
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }
}
template <typename T>
__device__ static inline 
thrust::complex<T> conj(thrust::complex<T>& in) {
    return thrust::conj(in);
}
template <typename T>
__global__ void cal_stress_mgga(
    const int spin,
    const int nrxx,
    const T w1,
    const thrust::complex<T> * gradwfc,
    T * crosstaus)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= nrxx) { return; }
    int ipol = 0;
    for (int ix = 0; ix < 3; ix++) {
        for (int iy = 0; iy < ix + 1; iy++) {
            crosstaus[spin * nrxx * 6 + ipol * nrxx + idx]
                += 2.0 * w1
                * (gradwfc[ix * nrxx + idx].real() * gradwfc[iy*nrxx + idx].real()
                +  gradwfc[ix * nrxx + idx].imag() * gradwfc[iy*nrxx + idx].imag());
            ipol += 1;
        }
    }
}

template <typename FPTYPE>
__global__ void cal_dbecp_noevc_nl(
        const int ipol,
        const int jpol,
        const int npw,
        const int npwx,
        const int ik,
        const FPTYPE tpiba,
        const FPTYPE *gcar,
        const FPTYPE *kvec_c,
        thrust::complex<FPTYPE> *vkbi,
        thrust::complex<FPTYPE> *vkbj,
        thrust::complex<FPTYPE> *vkb,
        thrust::complex<FPTYPE> *vkb1,
        thrust::complex<FPTYPE> *vkb2,
        thrust::complex<FPTYPE> *dbecp_noevc)
{
    int i = blockIdx.x;
    const thrust::complex<FPTYPE>* pvkb0i = vkbi + i * npwx;
    const thrust::complex<FPTYPE>* pvkb0j = vkbj + i * npwx;
    thrust::complex<FPTYPE>* pvkb = nullptr;
    thrust::complex<FPTYPE>* pdbecp_noevc = dbecp_noevc + i * npwx;
    // third term of dbecp_noevc
    //std::complex<FPTYPE>* pvkb = &vkb2(i,0);
    //std::complex<FPTYPE>* pdbecp_noevc = &dbecp_noevc(i, 0);
    FPTYPE qvec[3] = {0, 0, 0};
    for (int ig = threadIdx.x; ig < npw; ig += blockDim.x)
    {
        pvkb = vkb1 + i * npwx;
        qvec[ipol] = gcar[(ik * npwx + ig) * 3 + ipol] + kvec_c[ik * 3 + ipol];
        qvec[jpol] = gcar[(ik * npwx + ig) * 3 + jpol] + kvec_c[ik * 3 + jpol];
        pvkb[ig] += 0.5 * qvec[ipol] * pvkb0j[ig] +
                    0.5 * qvec[jpol] * pvkb0i[ig];
        pdbecp_noevc[ig] -= 2.0 * pvkb[ig];
        if (ipol == jpol) {
            pvkb = vkb + i * npwx;
            pdbecp_noevc[ig] -= pvkb[ig];
        }
        pvkb = vkb2 + i * npwx;
        for (int ii = 0; ii < 3; ii++) {
            qvec[ii] = gcar[(ik * npwx + ig) * 3 + ii] + kvec_c[ik * 3 + ii];
        }
        FPTYPE qvec_norm2 = qvec[0] * qvec[0] + qvec[1] * qvec[1] + qvec[2] * qvec[2];
        FPTYPE qm1 = qvec_norm2 > 1e-16 ? 1.0 / sqrt(qvec_norm2) : 0;
        pdbecp_noevc[ig] -= 2.0 * pvkb[ig] * qvec[ipol] *
                            qvec[jpol] * qm1 *	tpiba;
    } // end ig
}

template <typename FPTYPE>
__global__ void cal_stress_nl(
        const bool nondiagonal,
        const int ipol,
        const int jpol,
        const int nkb,
        const int ntype,
        const int spin,
        const int deeq_2,
        const int deeq_3,
        const int deeq_4,
        const int *atom_nh,
        const int *atom_na,
        const FPTYPE *d_wg,
        const bool occ,
        const FPTYPE* d_ekb,
        const FPTYPE* qq_nt,
        const FPTYPE *deeq,
        const thrust::complex<FPTYPE> *becp,
        const thrust::complex<FPTYPE> *dbecp,
        FPTYPE *stress)
{
    int ib = blockIdx.x / ntype;
    int it = blockIdx.x % ntype;

    int iat = 0;
    int sum = 0;
    for (int ii = 0; ii < it; ii++) {
        iat += atom_na[ii];
        sum += atom_na[ii] * atom_nh[ii];
    }

    FPTYPE stress_var = 0;
    FPTYPE fac;
    if (occ)
    {
        fac = d_wg[ib];
    }
    else
    {
        fac = d_wg[0];
    }
    FPTYPE ekb_now = 0.0;
    if (d_ekb != nullptr)
    {
        ekb_now = d_ekb[ib];
    }
    const int nproj = atom_nh[it];
    for (int ia = 0; ia < atom_na[it]; ia++)
    {
        for (int ii = threadIdx.x; ii < nproj * nproj; ii += blockDim.x) {
            const int ip1 = ii / nproj, ip2 = ii % nproj;
            if(!nondiagonal && ip1 != ip2) {
                continue;
            }
            FPTYPE ps_qq = 0;
            if (ekb_now != 0)
            {
                ps_qq = -ekb_now * qq_nt[it * deeq_3 * deeq_4 + ip1 * deeq_4 + ip2];
            }
            const FPTYPE ps = deeq[((spin * deeq_2 + iat) * deeq_3 + ip1) * deeq_4 + ip2] + ps_qq;
            const int inkb1 = sum + ip1;
            const int inkb2 = sum + ip2;
            //out<<"\n ps = "<<ps;
            const FPTYPE dbb = ( conj( dbecp[ ib * nkb + inkb1] ) * becp[ ib * nkb + inkb2] ).real();
            stress_var -= ps * fac * dbb;
        }
        ++iat;
        sum+=nproj;
    }//ia
    __syncwarp();
    warp_reduce(stress_var);
    if (threadIdx.x % WARP_SIZE == 0) {
        atomicAdd(stress + ipol * 3 + jpol, stress_var);
    }
}

template <typename FPTYPE>
__global__ void cal_multi_dot(const int npw,
                              const FPTYPE fac,
                              const FPTYPE* gk1,
                              const FPTYPE* gk2,
                              const FPTYPE* d_kfac,
                              const thrust::complex<FPTYPE>* psi,
                              FPTYPE* sum)
{
    __shared__ FPTYPE s_sum[THREADS_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheid = threadIdx.x;
    FPTYPE local_sum = 0;
    while (tid < npw) {
        local_sum += fac * gk1[tid] * gk2[tid] * d_kfac[tid] * thrust::norm(psi[tid]);
        tid += blockDim.x * gridDim.x;
    }
    s_sum[cacheid] = local_sum;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (cacheid < s) {
            s_sum[cacheid] += s_sum[cacheid + s];
        }
        __syncthreads();
    }
    if (cacheid == 0) {
        atomicAdd(sum, s_sum[0]);
    }
}

template <typename FPTYPE>
void cal_dbecp_noevc_nl_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* ctx,
                                                                        const int& ipol,
                                                                        const int& jpol,
                                                                        const int& nkb,
                                                                        const int& npw,
                                                                        const int& npwx,
                                                                        const int& ik,
                                                                        const FPTYPE& tpiba,
                                                                        const FPTYPE* gcar,
                                                                        const FPTYPE* kvec_c,
                                                                        std::complex<FPTYPE>* vkbi,
                                                                        std::complex<FPTYPE>* vkbj,
                                                                        std::complex<FPTYPE>* vkb,
                                                                        std::complex<FPTYPE>* vkb1,
                                                                        std::complex<FPTYPE>* vkb2,
                                                                        std::complex<FPTYPE>* dbecp_noevc)
{
    cal_dbecp_noevc_nl<FPTYPE><<<nkb, THREADS_PER_BLOCK>>>(
            ipol,
            jpol,
            npw,
            npwx,
            ik,
            tpiba,
            gcar,
            kvec_c,
            reinterpret_cast<thrust::complex<FPTYPE>*>(vkbi),
            reinterpret_cast<thrust::complex<FPTYPE>*>(vkbj),
            reinterpret_cast<thrust::complex<FPTYPE>*>(vkb),
            reinterpret_cast<thrust::complex<FPTYPE>*>(vkb1),
            reinterpret_cast<thrust::complex<FPTYPE>*>(vkb2),
            reinterpret_cast<thrust::complex<FPTYPE>*>(dbecp_noevc));

    cudaCheckOnDebug();
}

template <typename FPTYPE>
void cal_stress_nl_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* ctx,
                                                                   const bool& nondiagonal,
                                                                   const int& ipol,
                                                                   const int& jpol,
                                                                   const int& nkb,
                                                                   const int& nbands_occ,
                                                                   const int& ntype,
                                                                   const int& spin,
                                                                   const int& deeq_2,
                                                                   const int& deeq_3,
                                                                   const int& deeq_4,
                                                                   const int* atom_nh,
                                                                   const int* atom_na,
                                                                   const FPTYPE* d_wg,
                                                                   const bool& occ,
                                                                   const FPTYPE* d_ekb,
                                                                   const FPTYPE* qq_nt,
                                                                   const FPTYPE* deeq,
                                                                   const std::complex<FPTYPE>* becp,
                                                                   const std::complex<FPTYPE>* dbecp,
                                                                   FPTYPE* stress)
{
     cal_stress_nl<FPTYPE><<<nbands_occ * ntype, THREADS_PER_BLOCK>>>(
             nondiagonal,
             ipol,
             jpol,
             nkb,
             ntype,
             spin,
             deeq_2,
             deeq_3,
             deeq_4,
             atom_nh,
             atom_na,
             d_wg,
             occ,
             d_ekb,
             qq_nt,
             deeq,
             reinterpret_cast<const thrust::complex<FPTYPE>*>(becp),
             reinterpret_cast<const thrust::complex<FPTYPE>*>(dbecp),
             stress);// array of data

    cudaCheckOnDebug();
}

template <typename FPTYPE>
__global__ void cal_stress_nl(
        const int ipol,
        const int jpol,
        const int nkb,
        const int ntype,
        const int deeq_2,
        const int deeq_3,
        const int deeq_4,
        const int *atom_nh,
        const int *atom_na,
        const FPTYPE *d_wg,
        const bool occ,
        const FPTYPE* d_ekb,
        const FPTYPE* qq_nt,
        const thrust::complex<FPTYPE> *deeq_nc,
        const thrust::complex<FPTYPE> *becp,
        const thrust::complex<FPTYPE> *dbecp,
        FPTYPE *stress)
{
    const int ib = blockIdx.x / ntype; // index of loop-nbands
    const int ib2  = ib * 2;
    const int it = blockIdx.x % ntype; // index of loop-ntype

    int iat = 0; // calculate the begin of atomic index
    int sum = 0; // calculate the begin of atomic-orbital index
    for (int ii = 0; ii < it; ii++) {
        iat += atom_na[ii];
        sum += atom_na[ii] * atom_nh[ii];
    }

    FPTYPE stress_var = 0;
    FPTYPE fac;
    if (occ)
    {
        fac = d_wg[ib];
    }
    else
    {
        fac = d_wg[0];
    }
    FPTYPE ekb_now = 0.0;
    if (d_ekb != nullptr)
    {
        ekb_now = d_ekb[ib];
    }
    const int nproj = atom_nh[it];
    for (int ia = 0; ia < atom_na[it]; ia++)
    {
        for (int ii = threadIdx.x; ii < nproj * nproj; ii += blockDim.x) {
            const int ip1 = ii / nproj;
	        const int ip2 = ii % nproj;
            thrust::complex<FPTYPE> ps_qq = 0;
            if(ekb_now != 0)
            {
                ps_qq = thrust::complex<FPTYPE>(- ekb_now * qq_nt[it * deeq_3 * deeq_4 + ip1 * deeq_4 + ip2], 0.0);
            }
            const thrust::complex<FPTYPE> ps0 = deeq_nc[((iat + ia) * deeq_3 + ip1) * deeq_4 + ip2] + ps_qq;
            const thrust::complex<FPTYPE> ps1 = deeq_nc[((1 * deeq_2 + iat + ia) * deeq_3 + ip1) * deeq_4 + ip2];
            const thrust::complex<FPTYPE> ps2 = deeq_nc[((2 * deeq_2 + iat + ia) * deeq_3 + ip1) * deeq_4 + ip2];
            const thrust::complex<FPTYPE> ps3 = deeq_nc[((3 * deeq_2 + iat + ia) * deeq_3 + ip1) * deeq_4 + ip2] + ps_qq;
            const int inkb1 = sum + ip1;
            const int inkb2 = sum + ip2;
            //out<<"\n ps = "<<ps;
            const thrust::complex<FPTYPE> dbb0 = conj(dbecp[ib2 * nkb + inkb1]) * becp[ib2 * nkb + inkb2];
            const thrust::complex<FPTYPE> dbb1 = conj(dbecp[ib2 * nkb + inkb1]) * becp[(ib2+1) * nkb + inkb2];
            const thrust::complex<FPTYPE> dbb2 = conj(dbecp[(ib2+1) * nkb + inkb1]) * becp[ib2 * nkb + inkb2];
            const thrust::complex<FPTYPE> dbb3 = conj(dbecp[(ib2+1) * nkb + inkb1]) * becp[(ib2+1) * nkb + inkb2];
            stress_var -= fac * (ps0 * dbb0 + ps1 * dbb1 + ps2 * dbb2 + ps3 * dbb3).real();
        }
        ++iat;
        sum+=nproj;
    }//ia
    __syncwarp();
    warp_reduce(stress_var);
    if (threadIdx.x % WARP_SIZE == 0) {
        atomicAdd(stress + ipol * 3 + jpol, stress_var);
    }
}

template <typename FPTYPE>
void cal_stress_nl_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* ctx,
                                                                   const int& ipol,
                                                                   const int& jpol,
                                                                   const int& nkb,
                                                                   const int& nbands_occ,
                                                                   const int& ntype,
                                                                   const int& deeq_2,
                                                                   const int& deeq_3,
                                                                   const int& deeq_4,
                                                                   const int* atom_nh,
                                                                   const int* atom_na,
                                                                   const FPTYPE* d_wg,
                                                                   const bool& occ,
                                                                   const FPTYPE* d_ekb,
                                                                   const FPTYPE* qq_nt,
                                                                   const std::complex<FPTYPE>* deeq_nc,
                                                                   const std::complex<FPTYPE>* becp,
                                                                   const std::complex<FPTYPE>* dbecp,
                                                                   FPTYPE* stress)
{
     cal_stress_nl<FPTYPE><<<nbands_occ * ntype, THREADS_PER_BLOCK>>>(
             ipol,
             jpol,
             nkb,
             ntype,
             deeq_2,
             deeq_3,
             deeq_4,
             atom_nh,
             atom_na,
             d_wg,
             occ,
             d_ekb,
             qq_nt,
             reinterpret_cast<const thrust::complex<FPTYPE>*>(deeq_nc),
             reinterpret_cast<const thrust::complex<FPTYPE>*>(becp),
             reinterpret_cast<const thrust::complex<FPTYPE>*>(dbecp),
             stress);// array of data

    cudaCheckOnDebug();
}

template <typename FPTYPE>
FPTYPE cal_multi_dot_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const int& npw,
                                                                   const FPTYPE& fac,
                                                                   const FPTYPE* gk1,
                                                                   const FPTYPE* gk2,
                                                                   const FPTYPE* d_kfac,
                                                                   const std::complex<FPTYPE>* psi)
{
    FPTYPE* d_sum = nullptr;
    hipMalloc(&d_sum, sizeof(FPTYPE) * 1);
    hipMemset(d_sum, 0, sizeof(FPTYPE) * 1);
    int block = (npw + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cal_multi_dot<FPTYPE><<<block, THREADS_PER_BLOCK>>>(
        npw, fac, gk1, gk2, d_kfac, reinterpret_cast<const thrust::complex<FPTYPE>*>(psi), d_sum);
    FPTYPE sum;
    hipMemcpy(&sum, d_sum, sizeof(FPTYPE) * 1, hipMemcpyDeviceToHost);
    hipFree(d_sum);

    cudaCheckOnDebug();
    return sum;
}

template <typename T, typename Device>
void cal_stress_mgga_op<T, Device>::operator()(
    const int& spin,
    const int& nrxx,
    const Real& w1,
    const T * gradwfc,
    Real * crosstaus)
{
    auto gradwfc_ = reinterpret_cast<const thrust::complex<Real>*>(gradwfc);
    const int block = (nrxx + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cal_stress_mgga<Real><<<block, THREADS_PER_BLOCK>>>(
        spin, nrxx, w1, gradwfc_, crosstaus);

    cudaCheckOnDebug();
}




template <typename FPTYPE>
__global__ void cal_vkb(
    const int npw,
    const int* indexes,
    const FPTYPE* vqs_in,
    const FPTYPE* ylms_in,
    const thrust::complex<FPTYPE>* sk_in,
    const thrust::complex<FPTYPE>* pref_in,
    thrust::complex<FPTYPE>* vkbs_out
){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int ih =  blockIdx.y;

    thrust::complex<FPTYPE>* vkb_ptr = vkbs_out + ih * npw;
    const FPTYPE* ylm_ptr = ylms_in + indexes[ih*4] * npw;
    const FPTYPE* vq_ptr = vqs_in + indexes[ih*4+1] * npw;
    if(idx<npw) vkb_ptr[idx] = ylm_ptr[idx] * vq_ptr[idx] * sk_in[idx] * pref_in[ih];              
    
}

template <typename FPTYPE>
__global__ void cal_vkb_deri(
        const int npw,
        const int ipol,
        const int jpol,
        const int* indexes,
        const FPTYPE* vqs_in, const FPTYPE* vqs_deri_in,
        const FPTYPE* ylms_in, const FPTYPE* ylms_deri_in,
        const thrust::complex<FPTYPE>* sk_in,
        const thrust::complex<FPTYPE>* pref_in,
        const FPTYPE* gk_in,
        thrust::complex<FPTYPE>* vkbs_out
){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int ih =  blockIdx.y;

    thrust::complex<FPTYPE>* vkb_ptr = vkbs_out + ih * npw;
    const FPTYPE* ylm_ptr = ylms_in + indexes[ih*4] * npw;
    const FPTYPE* vq_ptr = vqs_in + indexes[ih*4 + 1] * npw;

    const FPTYPE* ylm_deri_ptr1 = ylms_deri_in + indexes[ih*4+2] * npw;
    const FPTYPE* ylm_deri_ptr2 = ylms_deri_in + indexes[ih*4+3] * npw;
    const FPTYPE* vq_deri_ptr = vqs_deri_in + indexes[ih*4+1] * npw;
    const FPTYPE* gkn = &gk_in[4 * npw];
    const FPTYPE* gk = &gk_in[idx * 3];

    if(idx<npw) {
        vkb_ptr[idx] = thrust::complex<FPTYPE>(0.0, 0.0);
        if(ipol == jpol)
        {
            vkb_ptr[idx] -= ylm_ptr[idx] * vq_ptr[idx] * sk_in[idx] * pref_in[ih];
        }
        vkb_ptr[idx] -= (gk[ipol] * ylm_deri_ptr2[idx] 
                        + gk[jpol] * ylm_deri_ptr1[idx]) 
                        * vq_ptr[idx] * sk_in[idx] * pref_in[ih];

        vkb_ptr[idx] -= 2.0 * ylm_ptr[idx] * vq_deri_ptr[idx] * sk_in[idx] * pref_in[ih]
                    * gk[ipol] * gk[jpol] * gkn[idx];  
    }
}


template <typename FPTYPE>
__global__ void cal_vq(
        const FPTYPE* tab,
        int it, const FPTYPE* gk, int npw,
        const int tab_2,const int tab_3,  const FPTYPE table_interval, 
        const int nbeta, FPTYPE* vq
){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int ib =  blockIdx.y;

    FPTYPE* vq_ptr = &vq[ib * npw];
    const FPTYPE* gnorm = &gk[3 * npw];
    if(idx<npw) vq_ptr[idx] = _polynomial_interpolation(
        tab, it, ib, tab_2, tab_3, table_interval, gnorm[idx]);
}

template <typename FPTYPE>
__global__ void cal_vq_deri(
        const FPTYPE* tab,
        int it, const FPTYPE* gk, int npw,
        const int tab_2,const int tab_3,  const FPTYPE table_interval, 
        const int nbeta, FPTYPE* vq
){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int ib =  blockIdx.y;

    FPTYPE* vq_ptr = &vq[ib * npw];
    const FPTYPE* gnorm = &gk[3 * npw];
    if(idx<npw) vq_ptr[idx] = _polynomial_interpolation_nl(
        tab, it, ib, tab_2, tab_3, table_interval, gnorm[idx]);
}

template <typename FPTYPE>
__global__ void cal_stress_drhoc_aux0(
        const FPTYPE* r, const FPTYPE* rhoc, 
        const FPTYPE *gx_arr, const FPTYPE *rab, FPTYPE *drhocg, 
        const int mesh, const int igl0, const int ngg, const double omega
){
    const double FOUR_PI =  4.0 * 3.14159265358979323846;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= ngg) {return;}

    FPTYPE rhocg1=0.0;
    FPTYPE gx = gx_arr[idx];

    auto aux = [](FPTYPE r, FPTYPE rhoc, FPTYPE gx, FPTYPE rab) -> FPTYPE{
        return r * rhoc * (r * cos (gx * r) / gx - sin (gx * r) / (gx * gx)) * rab;
    };

    FPTYPE f_0 = aux(r[0],rhoc[0], gx, rab[0]);
    for( int ir = 1 ; ir< mesh - 2; ir+=2)
    {
        rhocg1 += 2 * aux(r[ir],rhoc[ir], gx, rab[ir]) + aux(r[ir+1],rhoc[ir+1], gx, rab[ir+1]);
    }//ir
    FPTYPE f_2 = aux(r[mesh - 2],rhoc[mesh - 2], gx, rab[mesh - 2]);
    FPTYPE f_1 = aux(r[mesh - 1],rhoc[mesh - 1], gx, rab[mesh - 1]);

    rhocg1 += f_2+f_2;
    rhocg1 += rhocg1;
    rhocg1 += f_0 + f_1;
    rhocg1/=3.0;

    drhocg [idx] = FOUR_PI / omega * rhocg1;
}

template <typename FPTYPE>
__global__ void cal_stress_drhoc_aux1(
        const FPTYPE* r, const FPTYPE* rhoc, 
        const FPTYPE *gx_arr, const FPTYPE *rab, FPTYPE *drhocg, 
        const int mesh, const int igl0, const int ngg, const double omega
){
    const double FOUR_PI =  4.0 * 3.14159265358979323846;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= ngg) {return;}

    FPTYPE rhocg1=0.0;
    FPTYPE gx = gx_arr[idx];

    auto aux = [](FPTYPE r, FPTYPE rhoc, FPTYPE gx, FPTYPE rab) -> FPTYPE{
        return sin (gx * r) / (gx * r) * r * r * rhoc * rab;
    };

    FPTYPE f_0 = r[0] * r[0] * rhoc[0] * rab[0];
    for( int ir = 1 ; ir< mesh - 2; ir+=2)
    {
        rhocg1 += 2 * aux(r[ir],rhoc[ir], gx, rab[ir]) + aux(r[ir+1],rhoc[ir+1], gx, rab[ir+1]);
    }//ir
    
    FPTYPE f_2 = aux(r[mesh - 2],rhoc[mesh - 2], gx, rab[mesh - 2]);
    FPTYPE f_1 = aux(r[mesh - 1],rhoc[mesh - 1], gx, rab[mesh - 1]);

    rhocg1 += f_2+f_2;
    rhocg1 += rhocg1;
    rhocg1 += f_0 + f_1;
    rhocg1/=3.0;

    drhocg [idx] = FOUR_PI * rhocg1 / omega;
}


template <typename FPTYPE>
__global__ void cal_stress_drhoc_aux2(
        const FPTYPE* r, const FPTYPE* rhoc, 
        const FPTYPE *gx_arr, const FPTYPE *rab, FPTYPE *drhocg, 
        const int mesh, const int igl0, const int ngg, const double omega
){


    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= ngg) {return;}

    FPTYPE rhocg1=0.0;
    FPTYPE gx = gx_arr[idx];    

    auto aux = [](FPTYPE r, FPTYPE rhoc, FPTYPE gx, FPTYPE rab) -> FPTYPE{
        return r < 1.0e-8 ? rab * rhoc : rab * rhoc * sin(gx * r) / (gx * r);
    };


    FPTYPE f_0 = r[0] * r[0] * rhoc[0] * rab[0];
    for( int ir = 1 ; ir< mesh - 2; ir+=2)
    {
        rhocg1 += 2 * aux(r[ir],rhoc[ir], gx, rab[ir]) + aux(r[ir+1],rhoc[ir+1], gx, rab[ir+1]);
    }//ir
    FPTYPE f_2 = aux(r[mesh - 2],rhoc[mesh - 2], gx, rab[mesh - 2]);
    FPTYPE f_1 = aux(r[mesh - 1],rhoc[mesh - 1], gx, rab[mesh - 1]);
    
    rhocg1 += f_2+f_2;
    rhocg1 += rhocg1;
    rhocg1 += f_0 + f_1;
    rhocg1/=3.0;

    drhocg [idx] = rhocg1;
}


template <typename FPTYPE>
__global__ void cal_stress_drhoc_aux3(
        const FPTYPE* r, const FPTYPE* rhoc, 
        const FPTYPE *gx_arr, const FPTYPE *rab, FPTYPE *drhocg, 
        const int mesh, const int igl0, const int ngg, const double omega
){
    const double FOUR_PI =  4.0 * 3.14159265358979323846;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= ngg) {return;}

    FPTYPE rhocg1=0.0;
    FPTYPE gx = gx_arr[idx];    
    const FPTYPE pow_gx = gx * gx;

    auto aux = [](FPTYPE r, FPTYPE rhoc, FPTYPE gx, FPTYPE rab) -> FPTYPE{
        return rab * rhoc * (r * cos(gx * r)/gx - sin(gx * r)/(gx * gx));
    };

    FPTYPE f_0 = r[0] * r[0] * rhoc[0] * rab[0];
    for( int ir = 1 ; ir< mesh - 2; ir+=2)
    {
        rhocg1 += 2 * aux(r[ir],rhoc[ir], gx, rab[ir]) + aux(r[ir+1],rhoc[ir+1], gx, rab[ir+1]);
    }//ir
    FPTYPE f_2 = aux(r[mesh - 2],rhoc[mesh - 2], gx, rab[mesh - 2]);
    FPTYPE f_1 = aux(r[mesh - 1],rhoc[mesh - 1], gx, rab[mesh - 1]);
    
    rhocg1 += f_2+f_2;
    rhocg1 += rhocg1;
    rhocg1 += f_0 + f_1;
    rhocg1/=3.0;

    // calculations after Simpson Integral
    const double g2a = pow_gx / 4.0;
    rhocg1 *= FOUR_PI / omega / 2.0 / gx;
    rhocg1 += FOUR_PI / omega * gx_arr[ngg] * exp(-g2a) * (g2a + 1) / (pow_gx*pow_gx);
    drhocg [idx] = rhocg1;
}



template <typename FPTYPE>
__global__ void cal_force_npw(
        const thrust::complex<FPTYPE> *psiv,
        const FPTYPE* gv_x, const FPTYPE* gv_y, const FPTYPE* gv_z,
        const FPTYPE* rhocgigg_vec,
        FPTYPE* force,
        const FPTYPE pos_x, const FPTYPE pos_y, const FPTYPE pos_z,
        const int npw,
        const FPTYPE omega, const FPTYPE tpiba
){
    const double TWO_PI = 2.0 * 3.14159265358979323846;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int begin_idx = tid * 1024;
    if(begin_idx > npw) return;

    FPTYPE t_force0 = 0;
    FPTYPE t_force1 = 0;
    FPTYPE t_force2 = 0;
    for(int ig = begin_idx; ig<begin_idx+1024 && ig<npw;ig++) {
        const thrust::complex<FPTYPE> psiv_conj = conj(psiv[ig]);

        const FPTYPE arg = TWO_PI * (gv_x[ig] * pos_x + gv_y[ig] * pos_y + gv_z[ig] * pos_z);
        const FPTYPE sinp = sin(arg);
        const FPTYPE cosp = cos(arg);
        const thrust::complex<FPTYPE> expiarg = thrust::complex<FPTYPE>(sinp, cosp);

        const thrust::complex<FPTYPE> tmp_var = psiv_conj * expiarg * tpiba * omega * rhocgigg_vec[ig];

        const thrust::complex<FPTYPE> ipol0 = tmp_var * gv_x[ig];
        t_force0 += ipol0.real();

        const thrust::complex<FPTYPE> ipol1 = tmp_var * gv_y[ig];
        t_force1 += ipol1.real();

        const thrust::complex<FPTYPE> ipol2 = tmp_var * gv_z[ig];
        t_force2 += ipol2.real();
    }
    atomicAdd(&force[0], t_force0);
    atomicAdd(&force[1], t_force1);
    atomicAdd(&force[2], t_force2);
}

template <typename FPTYPE>
void cal_vkb_op<FPTYPE, base_device::DEVICE_GPU>::operator()(
        const base_device::DEVICE_GPU* ctx,
        const int nh,
        const int npw,
        const int* indexes,
        const FPTYPE* vqs_in,
        const FPTYPE* ylms_in,
        const std::complex<FPTYPE>* sk_in,
        const std::complex<FPTYPE>* pref_in,
        std::complex<FPTYPE>* vkbs_out
    )
{
    const int block = (npw + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 gridsize(block,nh);

    cal_vkb<FPTYPE><<<gridsize,THREADS_PER_BLOCK>>>(
        npw, indexes, vqs_in, ylms_in,
        reinterpret_cast<const thrust::complex<FPTYPE>*>(sk_in), 
        reinterpret_cast<const thrust::complex<FPTYPE>*>(pref_in), 
        reinterpret_cast<thrust::complex<FPTYPE>*>(vkbs_out)
        
    );

}

template <typename FPTYPE>
void cal_vkb_deri_op<FPTYPE, base_device::DEVICE_GPU>::operator()(
        const base_device::DEVICE_GPU* ctx,
        const int nh,
        const int npw,
        const int ipol,
        const int jpol,
        const int* indexes,
        const FPTYPE* vqs_in,
        const FPTYPE* vqs_deri_in,
        const FPTYPE* ylms_in,
        const FPTYPE* ylms_deri_in,
        const std::complex<FPTYPE>* sk_in,
        const std::complex<FPTYPE>* pref_in,
        const FPTYPE* gk_in,
        std::complex<FPTYPE>* vkbs_out)
{
    const int block = (npw + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 gridsize(block,nh);

    cal_vkb_deri<FPTYPE><<<gridsize,THREADS_PER_BLOCK>>>(
        npw, ipol, jpol, indexes,
        vqs_in, vqs_deri_in, ylms_in, ylms_deri_in,
        reinterpret_cast<const thrust::complex<FPTYPE>*>(sk_in), 
        reinterpret_cast<const thrust::complex<FPTYPE>*>(pref_in),       
        gk_in,
        reinterpret_cast<thrust::complex<FPTYPE>*>(vkbs_out)
    );
}

template <typename FPTYPE>
void cal_vq_op<FPTYPE, base_device::DEVICE_GPU>::operator()(
        const base_device::DEVICE_GPU *ctx,
        const FPTYPE* tab,
        int it, const FPTYPE* gk, int npw,
        const int tab_2, const int tab_3, const FPTYPE table_interval, 
        const int nbeta, FPTYPE* vq
    )
{
    const int block = (npw + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 gridsize(block,nbeta);

    cal_vq<FPTYPE><<<gridsize,THREADS_PER_BLOCK>>>(
        tab, it, gk, npw, tab_2, tab_3,
        table_interval, nbeta, vq
    );
}


template <typename FPTYPE>
void cal_vq_deri_op<FPTYPE, base_device::DEVICE_GPU>::operator()(
        const base_device::DEVICE_GPU *ctx,
        const FPTYPE* tab,
        int it, const FPTYPE* gk, int npw,
        const int tab_2, const int tab_3, const FPTYPE table_interval, 
        const int nbeta, FPTYPE* vq
    )
{
    const int block = (npw + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    dim3 gridsize(block,nbeta);

    cal_vq_deri<FPTYPE><<<gridsize,THREADS_PER_BLOCK>>>(
        tab, it, gk, npw, tab_2, tab_3,
        table_interval, nbeta, vq
    );

    return ;
}


/**
 * The implementation of this operator is detailed in stress_op.h.
 * The function is called by the module as follows
 *      Type = 0 -> stress_cc
 *      Type = 1 -> stress_cc, force_cc
 *      Type = 2 -> force_scc
 *      Type = 3 -> stress_loc
 */
template <typename FPTYPE>
void cal_stress_drhoc_aux_op<FPTYPE, base_device::DEVICE_GPU>::operator()(
        const FPTYPE* r, const FPTYPE* rhoc,  
        const FPTYPE *gx_arr, const FPTYPE *rab, FPTYPE *drhocg, 
        const int mesh, const int igl0, const int ngg, const double omega,
        int type
    )
{
    const int block = (ngg + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    if(type == 0) {
        cal_stress_drhoc_aux0<FPTYPE><<<block,THREADS_PER_BLOCK>>>(
            r,rhoc,gx_arr,rab,drhocg,mesh,igl0,ngg,omega
        );
    } else if(type == 1 ){
        cal_stress_drhoc_aux1<FPTYPE><<<block,THREADS_PER_BLOCK>>>(
            r,rhoc,gx_arr,rab,drhocg,mesh,igl0,ngg,omega
        );        
    } else if(type == 2 ){
        cal_stress_drhoc_aux2<FPTYPE><<<block,THREADS_PER_BLOCK>>>(
            r,rhoc,gx_arr,rab,drhocg,mesh,igl0,ngg,omega
        );        
    } else if(type == 3 ){
        cal_stress_drhoc_aux3<FPTYPE><<<block,THREADS_PER_BLOCK>>>(
            r,rhoc,gx_arr,rab,drhocg,mesh,igl0,ngg,omega
        );        
    }
    return ;
}


template <typename FPTYPE>
void cal_force_npw_op<FPTYPE, base_device::DEVICE_GPU>::operator()(
        const std::complex<FPTYPE> *psiv,
        const FPTYPE* gv_x, const FPTYPE* gv_y, const FPTYPE* gv_z,
        const FPTYPE* rhocgigg_vec,
        FPTYPE* force,
        const FPTYPE pos_x, const FPTYPE pos_y, const FPTYPE pos_z,
        const int npw,
        const FPTYPE omega, const FPTYPE tpiba
    )
{
    // Divide the npw size range into blocksize 1024 blocks
    int t_size = 1024;
    int t_num = (npw%t_size) ? (npw/t_size + 1) : (npw/t_size);
    dim3 npwgrid(((t_num%THREADS_PER_BLOCK) ? (t_num/THREADS_PER_BLOCK + 1) : (t_num/THREADS_PER_BLOCK)));

    cal_force_npw <<< npwgrid, THREADS_PER_BLOCK >>> (
        reinterpret_cast<const thrust::complex<FPTYPE>*>(psiv),
        gv_x, gv_y, gv_z, rhocgigg_vec, force, pos_x, pos_y, pos_z,
        npw, omega, tpiba
    ); 
    return ;
}


template <>
void pointer_array_malloc<base_device::DEVICE_GPU>::operator()(
    void **ptr,
    const int n
)
{
    cudaErrcheck(hipMalloc(ptr, n * sizeof(void*)));
}

template struct pointer_array_malloc<base_device::DEVICE_GPU>;

template <>
void synchronize_ptrs<base_device::DEVICE_GPU>::operator()(
    void **ptr_out,
    const void **ptr_in,
    const int size)
{
    hipMemcpy(ptr_out, ptr_in, sizeof(void*) * size, hipMemcpyHostToDevice);
}

template <typename FPTYPE>
__global__ void cal_stress_onsite(
        const int nkb,
        const int ntype,
        const int wg_nc,
        const int ik,
        const int *atom_nh,
        const int *atom_na,
        const FPTYPE *d_wg,
        const thrust::complex<FPTYPE> *vu,
        const int* orbital_corr,
        const thrust::complex<FPTYPE> *becp,
        const thrust::complex<FPTYPE> *dbecp,
        FPTYPE *stress)
{
    const int ib = blockIdx.x / ntype; // index of loop-nbands
    const int ib2  = ib * 2;
    const int it = blockIdx.x % ntype; // index of loop-ntype
    if(orbital_corr[it] == -1) return;
    const int orbital_l = orbital_corr[it];
    const int ip_begin = orbital_l * orbital_l;
    const int tlp1 = 2 * orbital_l + 1;
    const int tlp1_2 = tlp1 * tlp1;

    int iat = 0; // calculate the begin of atomic index
    int sum = 0; // calculate the begin of atomic-orbital index
    for (int ii = 0; ii < it; ii++) {
        iat += atom_na[ii];
        sum += atom_na[ii] * atom_nh[ii];
        vu += 4 * tlp1_2 * atom_na[ii];// step for vu
    }

    FPTYPE stress_var = 0;
    const FPTYPE fac = d_wg[ik * wg_nc + ib];
    const int nprojs = atom_nh[it];
    for (int ia = 0; ia < atom_na[it]; ia++)
    {
        for (int mm = threadIdx.x; mm < tlp1_2; mm += blockDim.x) {
            const int m1 = mm / tlp1;
            const int m2 = mm % tlp1;
            const int ip1 = ip_begin + m1;
            const int ip2 = ip_begin + m2;
            const int inkb1 = sum + ip1 + ib2 * nkb;
            const int inkb2 = sum + ip2 + ib2 * nkb;
            thrust::complex<FPTYPE> ps[4] = {vu[mm], vu[mm + tlp1_2], vu[mm + 2 * tlp1_2], vu[mm + 3 * tlp1_2]};
            //out<<"\n ps = "<<ps;
            const thrust::complex<FPTYPE> dbb0 = conj(dbecp[inkb1]) * becp[inkb2];
            const thrust::complex<FPTYPE> dbb1 = conj(dbecp[inkb1]) * becp[inkb2 + nkb];
            const thrust::complex<FPTYPE> dbb2 = conj(dbecp[inkb1 + nkb]) * becp[inkb2];
            const thrust::complex<FPTYPE> dbb3 = conj(dbecp[inkb1 + nkb]) * becp[inkb2 + nkb];
            stress_var -= fac * (ps[0] * dbb0 + ps[1] * dbb1 + ps[2] * dbb2 + ps[3] * dbb3).real();
        }
        ++iat;
        sum+=nprojs;
        vu += 4 * tlp1_2;
    }//ia
    __syncwarp();
    warp_reduce(stress_var);
    if (threadIdx.x % WARP_SIZE == 0) {
        atomicAdd(stress, stress_var);
    }
}

template <typename FPTYPE>
__global__ void cal_stress_onsite(
        const int nkb,
        const int ntype,
        const int wg_nc,
        const int ik,
        const int *atom_nh,
        const int *atom_na,
        const FPTYPE *d_wg,
        const double* lambda,
        const thrust::complex<FPTYPE> *becp,
        const thrust::complex<FPTYPE> *dbecp,
        FPTYPE *stress)
{
    const int ib = blockIdx.x / ntype; // index of loop-nbands
    const int ib2  = ib * 2;
    const int it = blockIdx.x % ntype; // index of loop-ntype

    int iat = 0; // calculate the begin of atomic index
    int sum = 0; // calculate the begin of atomic-orbital index
    for (int ii = 0; ii < it; ii++) {
        iat += atom_na[ii];
        sum += atom_na[ii] * atom_nh[ii];
    }

    FPTYPE stress_var = 0;
    const FPTYPE fac = d_wg[ik * wg_nc + ib];
    const int nprojs = atom_nh[it];
    for (int ia = 0; ia < atom_na[it]; ia++)
    {
        const thrust::complex<FPTYPE> coefficients0(lambda[iat*3+2], 0.0);
        const thrust::complex<FPTYPE> coefficients1(lambda[iat*3] , lambda[iat*3+1]);
        const thrust::complex<FPTYPE> coefficients2(lambda[iat*3] , -1 * lambda[iat*3+1]);
        const thrust::complex<FPTYPE> coefficients3(-1 * lambda[iat*3+2], 0.0);
        for (int ip = threadIdx.x; ip < nprojs; ip += blockDim.x) {
            const int inkb = sum + ip + ib2 * nkb;
            //out<<"\n ps = "<<ps;
            const thrust::complex<FPTYPE> dbb0 = conj(dbecp[inkb]) * becp[inkb];
            const thrust::complex<FPTYPE> dbb1 = conj(dbecp[inkb]) * becp[inkb + nkb];
            const thrust::complex<FPTYPE> dbb2 = conj(dbecp[inkb + nkb]) * becp[inkb];
            const thrust::complex<FPTYPE> dbb3 = conj(dbecp[inkb + nkb]) * becp[inkb + nkb];
            stress_var -= fac * (coefficients0 * dbb0 + coefficients1 * dbb1 + coefficients2 * dbb2 + coefficients3 * dbb3).real();
        }
        ++iat;
        sum+=nprojs;
    }//ia
    __syncwarp();
    warp_reduce(stress_var);
    if (threadIdx.x % WARP_SIZE == 0) {
        atomicAdd(stress, stress_var);
    }
}

//kernel for DFTU stress
template <typename FPTYPE>
void cal_stress_nl_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* ctx,
                    const int& nkb,
                    const int& nbands_occ,
                    const int& ntype,
                    const int& wg_nc,
                    const int& ik,
                    const int* atom_nh,
                    const int* atom_na,
                    const FPTYPE* d_wg,
                    const std::complex<FPTYPE>* vu,
                    const int* orbital_corr,
                    const std::complex<FPTYPE>* becp,
                    const std::complex<FPTYPE>* dbecp,
                    FPTYPE* stress)
{
    cal_stress_onsite<FPTYPE><<<nbands_occ * ntype, THREADS_PER_BLOCK>>>(
             nkb,
             ntype,
             wg_nc,
             ik,
             atom_nh,
             atom_na,
             d_wg,
             reinterpret_cast<const thrust::complex<FPTYPE>*>(vu),
             orbital_corr,
             reinterpret_cast<const thrust::complex<FPTYPE>*>(becp),
             reinterpret_cast<const thrust::complex<FPTYPE>*>(dbecp),
             stress);// array of data

    cudaCheckOnDebug();
}
// kernel for DeltaSpin stress
template <typename FPTYPE>
void cal_stress_nl_op<FPTYPE, base_device::DEVICE_GPU>::operator()(const base_device::DEVICE_GPU* ctx,
                    const int& nkb,
                    const int& nbands_occ,
                    const int& ntype,
                    const int& wg_nc,
                    const int& ik,
                    const int* atom_nh,
                    const int* atom_na,
                    const FPTYPE* d_wg,
                    const double* lambda,
                    const std::complex<FPTYPE>* becp,
                    const std::complex<FPTYPE>* dbecp,
                    FPTYPE* stress)
{
    cal_stress_onsite<FPTYPE><<<nbands_occ * ntype, THREADS_PER_BLOCK>>>(
             nkb,
             ntype,
             wg_nc,
             ik,
             atom_nh,
             atom_na,
             d_wg,
             lambda,
             reinterpret_cast<const thrust::complex<FPTYPE>*>(becp),
             reinterpret_cast<const thrust::complex<FPTYPE>*>(dbecp),
             stress);// array of data

    cudaCheckOnDebug();
}

template struct synchronize_ptrs<base_device::DEVICE_GPU>;

template struct cal_stress_mgga_op<std::complex<float>, base_device::DEVICE_GPU>;
template struct cal_stress_mgga_op<std::complex<double>, base_device::DEVICE_GPU>;

template struct cal_dbecp_noevc_nl_op<float, base_device::DEVICE_GPU>;
template struct cal_dbecp_noevc_nl_op<double, base_device::DEVICE_GPU>;

template struct cal_stress_nl_op<float, base_device::DEVICE_GPU>;
template struct cal_stress_nl_op<double, base_device::DEVICE_GPU>;


template struct cal_vq_op<double, base_device::DEVICE_GPU>;
template struct cal_vq_op<float, base_device::DEVICE_GPU>;

template struct cal_vq_deri_op<double, base_device::DEVICE_GPU>;
template struct cal_vq_deri_op<float, base_device::DEVICE_GPU>;

template struct cal_vkb_op<double, base_device::DEVICE_GPU>;
template struct cal_vkb_op<float, base_device::DEVICE_GPU>;

template struct cal_vkb_deri_op<double, base_device::DEVICE_GPU>;
template struct cal_vkb_deri_op<float, base_device::DEVICE_GPU>;

template struct cal_stress_drhoc_aux_op<double, base_device::DEVICE_GPU>;
template struct cal_stress_drhoc_aux_op<float, base_device::DEVICE_GPU>;

template struct cal_force_npw_op<double, base_device::DEVICE_GPU>;
template struct cal_force_npw_op<float, base_device::DEVICE_GPU>;

template struct cal_multi_dot_op<double, base_device::DEVICE_GPU>;
template struct cal_multi_dot_op<float, base_device::DEVICE_GPU>;

// template struct prepare_vkb_deri_ptr_op<double, base_device::DEVICE_GPU>;
// template struct prepare_vkb_deri_ptr_op<float, base_device::DEVICE_GPU>;
}  // namespace hamilt